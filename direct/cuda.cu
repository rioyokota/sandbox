#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <papi.h>
#include <sys/time.h>
#include <xmmintrin.h>

#define THREADS 512

double get_time() {
  struct timeval tv;
  hipDeviceSynchronize();
  gettimeofday(&tv,NULL);
  return (double)(tv.tv_sec+tv.tv_usec*1e-6);
}

__global__ void GPUkernel(int N, float * x, float * y, float * z, float * m,
			  float * p, float * ax, float * ay, float * az, float eps2) {
  int i = blockIdx.x * THREADS + threadIdx.x;
  float pi = 0;
  float axi = 0;
  float ayi = 0;
  float azi = 0;
  float xi = x[i];
  float yi = y[i];
  float zi = z[i];
  __shared__ float xj[THREADS], yj[THREADS], zj[THREADS], mj[THREADS];
  for ( int jb=0; jb<N/THREADS; jb++ ) {
    __syncthreads();
    xj[threadIdx.x] = x[jb*THREADS+threadIdx.x];
    yj[threadIdx.x] = y[jb*THREADS+threadIdx.x];
    zj[threadIdx.x] = z[jb*THREADS+threadIdx.x];
    mj[threadIdx.x] = m[jb*THREADS+threadIdx.x];
    __syncthreads();
    for( int j=0; j<THREADS; j++ ) {
      float dx = xj[j] - xi;
      float dy = yj[j] - yi;
      float dz = zj[j] - zi;
      float R2 = dx * dx + dy * dy + dz * dz + eps2;
      float invR = rsqrtf(R2);
      pi += mj[j] * invR;
      float invR3 = mj[j] * invR * invR * invR;
      axi += dx * invR3;
      ayi += dy * invR3;
      azi += dz * invR3;
    }
  }
  p[i] = pi;
  ax[i] = axi;
  ay[i] = ayi;
  az[i] = azi;
}

int main() {
// Initialize
  int N = 1 << 16;
  int i, j;
  float OPS = 20. * N * N * 1e-9;
  float EPS2 = 1e-6;
  double tic, toc;
  float * x = (float*) malloc(N * sizeof(float));
  float * y = (float*) malloc(N * sizeof(float));
  float * z = (float*) malloc(N * sizeof(float));
  float * m = (float*) malloc(N * sizeof(float));
  float * p = (float*) malloc(N * sizeof(float));
  float * ax = (float*) malloc(N * sizeof(float));
  float * ay = (float*) malloc(N * sizeof(float));
  float * az = (float*) malloc(N * sizeof(float));
  for (i=0; i<N; i++) {
    x[i] = drand48();
    y[i] = drand48();
    z[i] = drand48();
    m[i] = drand48() / N;
  }
  int Events[3] = {PAPI_L2_DCM, PAPI_L2_DCA, PAPI_TLB_DM};
  int EventSet = PAPI_NULL;
  long long values[3] = {0, 0, 0};
  PAPI_library_init(PAPI_VER_CURRENT);
  PAPI_create_eventset(&EventSet);
  PAPI_add_events(EventSet, Events, 3);
  printf("N      : %d\n",N);

// CUDA
  tic = get_time();
  float *x_d, *y_d, *z_d, *m_d, *p_d, *ax_d, *ay_d, *az_d;
  hipMalloc((void**)&x_d, N * sizeof(float));
  hipMalloc((void**)&y_d, N * sizeof(float));
  hipMalloc((void**)&z_d, N * sizeof(float));
  hipMalloc((void**)&m_d, N * sizeof(float));
  hipMalloc((void**)&p_d, N * sizeof(float));
  hipMalloc((void**)&ax_d, N * sizeof(float));
  hipMalloc((void**)&ay_d, N * sizeof(float));
  hipMalloc((void**)&az_d, N * sizeof(float));
  toc = get_time();
  //printf("malloc : %e s\n",toc-tic);
  tic = get_time();
  hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(z_d, z, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(m_d, m, N * sizeof(float), hipMemcpyHostToDevice);
  toc = get_time();
  //printf("memcpy : %e s\n",toc-tic);
  PAPI_start(EventSet);
  tic = get_time();
  GPUkernel<<<N/THREADS,THREADS>>>(N, x_d, y_d, z_d, m_d, p_d, ax_d, ay_d, az_d, EPS2);
  toc = get_time();
  PAPI_stop(EventSet,values);
  printf("L2 Miss: %lld L2 Access: %lld TLB Miss: %lld\n",values[0],values[1],values[2]);
  printf("CUDA   : %e s : %lf GFlops\n",toc-tic, OPS/(toc-tic));
  tic = get_time();
  hipMemcpy(p, p_d, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(ax, ax_d, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(ay, ay_d, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(az, az_d, N * sizeof(float), hipMemcpyDeviceToHost);
  toc = get_time();
  //printf("memcpy : %e s\n",toc-tic);
  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(m_d);
  hipFree(p_d);
  hipFree(ax_d);
  hipFree(ay_d);
  hipFree(az_d);
  for (i=0; i<3; i++) values[i] = 0;

// No CUDA
  float pdiff = 0, pnorm = 0, adiff = 0, anorm = 0;
  PAPI_start(EventSet);
  tic = get_time();
#pragma omp parallel for private(j)
  for (i=0; i<N; i++) {
    float pi = 0;
    float axi = 0;
    float ayi = 0;
    float azi = 0;
    float xi = x[i];
    float yi = y[i];
    float zi = z[i];
    for (j=0; j<N; j++) {
      float dx = x[j] - xi;
      float dy = y[j] - yi;
      float dz = z[j] - zi;
      float R2 = dx * dx + dy * dy + dz * dz + EPS2;
      float invR = 1.0f / sqrtf(R2);
      float invR3 = m[j] * invR * invR * invR;
      pi += m[j] * invR;
      axi += dx * invR3;
      ayi += dy * invR3;
      azi += dz * invR3;
    }
    pdiff += (p[i] - pi) * (p[i] - pi);
    pnorm += pi * pi;
    adiff += (ax[i] - axi) * (ax[i] - axi)
      + (ay[i] - ayi) * (ay[i] - ayi)
      + (az[i] - azi) * (az[i] - azi);
    anorm += axi * axi + ayi * ayi + azi * azi;    
  }
  toc = get_time();
  PAPI_stop(EventSet,values);
  printf("L2 Miss: %lld L2 Access: %lld TLB Miss: %lld\n",values[0],values[1],values[2]);
  printf("No CUDA: %e s : %lf GFlops\n",toc-tic, OPS/(toc-tic));
  printf("P ERR  : %e\n",sqrt(pdiff/pnorm));
  printf("A ERR  : %e\n",sqrt(adiff/anorm));

// DEALLOCATE
  free(x);
  free(y);
  free(z);
  free(m);
  free(p);
  free(ax);
  free(ay);
  free(az);
  return 0;
}


#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

int main(int argc, const char **argv) {
  int n = 1 << 20;
  int Nt = 10;
  float alpha = 1.0;
  float *x, *y;
  hipMallocManaged(&x, n * sizeof(float));
  hipMallocManaged(&y, n * sizeof(float));
  for (int i=0; i<n; i++)
    x[i] = drand48();
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSaxpy(cublas_handle,
		n,
		&alpha,
		x,
		1,
		y,
		1);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = 2 * int64_t(n);
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  printf("Saxpy: %.2f GFlops\n", cublas_flops);
  hipFree(x);
  hipFree(y);
  hipblasDestroy(cublas_handle);
}

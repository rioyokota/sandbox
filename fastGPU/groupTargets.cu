#include "hip/hip_runtime.h"
#include "Treecode.h"

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

namespace groupTargets
{

  template<typename T>
  static __global__ void shuffle(const int n, const int *map, const T *in, T *out)
  {
    const int gidx = blockDim.x*blockIdx.x + threadIdx.x;
    if (gidx >= n) return;
    out[gidx] = in[map[gidx]];
  }

  template<int NBITS>
  static __device__ unsigned long long getHilbert(int3 crd) {
    int i,xi, yi, zi;
    int mask;
    unsigned long long key;
    const int C[8] = {0, 1, 7, 6, 3, 2, 4, 5};

    int temp;

    mask = 1 << (NBITS - 1);
    key  = 0;

#pragma unroll
    for(i = 0; i < NBITS; i++, mask >>= 1)
      {
        xi = (crd.x & mask) ? 1 : 0;
        yi = (crd.y & mask) ? 1 : 0;
        zi = (crd.z & mask) ? 1 : 0;        

        const int index = (xi << 2) + (yi << 1) + zi;

        int Cvalue;
        if(index == 0)
	  {
	    temp = crd.z; crd.z = crd.y; crd.y = temp;
	    Cvalue = C[0];
	  }
        else  if(index == 1 || index == 5)
	  {
	    temp = crd.x; crd.x = crd.y; crd.y = temp;
	    if (index == 1) Cvalue = C[1];
	    else            Cvalue = C[5];
	  }
        else  if(index == 4 || index == 6)
	  {
	    crd.x = (crd.x) ^ (-1);
	    crd.z = (crd.z) ^ (-1);
	    if (index == 4) Cvalue = C[4];
	    else            Cvalue = C[6];
	  }
        else  if(index == 7 || index == 3)
	  {
	    temp  = (crd.x) ^ (-1);         
	    crd.x = (crd.y) ^ (-1);
	    crd.y = temp;
	    if (index == 3) Cvalue = C[3];
	    else            Cvalue = C[7];
	  }
        else
	  {
	    temp = (crd.z) ^ (-1);         
	    crd.z = (crd.y) ^ (-1);
	    crd.y = temp;          
	    Cvalue = C[2];
	  }   

        key = (key<<3) + Cvalue;
      } //end for

    return key;
  }

  template<int NBINS>
  static __global__ 
  void computeKeys(
		   const int n,
		   const float4 *d_domain,
		   const float4 *bodyPos,
		   unsigned long long *keys,
		   int *values)
  {
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= n) return;

    const float4 body = bodyPos[idx];

    const float4 domain = d_domain[0];
    const float inv_domain_size = 0.5f / domain.w;
    const float3 bmin = {domain.x - domain.w,
			 domain.y - domain.w,
			 domain.z - domain.w};

    const int xc = static_cast<int>((body.x - bmin.x) * inv_domain_size * (1<<NBINS));
    const int yc = static_cast<int>((body.y - bmin.y) * inv_domain_size * (1<<NBINS));
    const int zc = static_cast<int>((body.z - bmin.z) * inv_domain_size * (1<<NBINS));

    keys  [idx] = getHilbert<NBINS>(make_int3(xc,yc,zc));
    values[idx] = idx;
  }

  static __global__
  void mask_keys(
		 const int n, 
		 const unsigned long long mask,
		 unsigned long long *keys,
		 unsigned long long *keys_inv,
		 int *bodyBegIdx,
		 int *bodyEndIdx)
  {
    const int gidx = blockIdx.x*blockDim.x + threadIdx.x;
    if (gidx >= n) return;

    keys[gidx] &= mask;
    keys_inv[n-gidx-1] = keys[gidx];

    extern __shared__ unsigned long long shKeys[];

    const int tid = threadIdx.x;
    shKeys[tid+1] = keys[gidx] & mask;

    int shIdx = 0;
    int gmIdx = max(blockIdx.x*blockDim.x-1,0);
    if (tid == 1)
      {
        shIdx = blockDim.x+1;
        gmIdx = min(blockIdx.x*blockDim.x + blockDim.x,n-1);
      }
    if (tid < 2)
      shKeys[shIdx] = keys[gmIdx] & mask;

    __syncthreads();

    const int idx = tid+1;
    const unsigned long long currKey = shKeys[idx  ];
    const unsigned long long prevKey = shKeys[idx-1];
    const unsigned long long nextKey = shKeys[idx+1];

    if (currKey != prevKey || gidx == 0)
      bodyBegIdx[gidx] = gidx;
    else
      bodyBegIdx[gidx] = 0;

    if (currKey != nextKey || gidx == n-1)
      bodyEndIdx[n-1-gidx] = gidx+1;
    else
      bodyEndIdx[n-1-gidx] = 0;

  }

  __device__ unsigned int groupCounter= 0;

  static __global__
  void make_groups(const int n, const int NCRIT,
		   const int *bodyBegIdx, 
		   const int *bodyEndIdx,
		   int2 *targetCells)
  {
    const int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (gidx >= n) return;

    const int bodyBeg = bodyBegIdx[gidx];
    assert(gidx >= bodyBeg);

    const int igroup   = (gidx - bodyBeg)/NCRIT;
    const int groupBeg = bodyBeg + igroup * NCRIT;

    if (gidx == groupBeg)
      {
        const int groupIdx = atomicAdd(&groupCounter,1);
        const int bodyEnd = bodyEndIdx[n-1-gidx];
        targetCells[groupIdx] = make_int2(groupBeg, min(NCRIT, bodyEnd - groupBeg));
      }
  }

  struct keyCompare
  {
    __host__ __device__
    bool operator()(const unsigned long long x, const unsigned long long y)
    {
      return x < y;
    }
  };

};

void Treecode::groupTargets(float4 * d_domain, int2 * d_targetCells, int levelSplit, const int NCRIT)
{
  const int nthread = 256;
  cuda_mem<int> d_key, d_value;

  d_key.alloc(2.0*numBodies);
  d_value.alloc(numBodies);

  unsigned long long *d_keys = (unsigned long long*)d_key.ptr;
  int *d_values = d_value.ptr;

  numTargets = 0;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(groupTargets::groupCounter), &numTargets, sizeof(int)));

  const int nblock  = (numBodies-1)/nthread + 1;
  const int NBINS = 21; 

  hipDeviceSynchronize();
  const double t0 = get_time();
  groupTargets::computeKeys<NBINS><<<nblock,nthread>>>(numBodies, d_domain, d_bodyPos, d_keys, d_values);

  levelSplit = std::max(1,levelSplit);  /* pick the coarse segment boundaries at the levelSplit */
  unsigned long long mask= 0;
  for (int i = 0; i < NBINS; i++)
    {
      mask <<= 3;
      if (i < levelSplit)
	mask |= 0x7;
    }

  /* sort particles by PH key */
  thrust::device_ptr<unsigned long long> keys_beg(d_keys);
  thrust::device_ptr<unsigned long long> keys_end(d_keys + numBodies);
  thrust::device_ptr<int> vals_beg(d_value.ptr);
#if 1
  thrust::sort_by_key(keys_beg, keys_end, vals_beg); 
#else
  thrust::sort_by_key(keys_beg, keys_end, vals_beg, groupTargets::keyCompare());
#endif

#if 1
  groupTargets::shuffle<float4><<<nblock,nthread>>>(numBodies, d_value, d_bodyPos, d_bodyPos2);

  cuda_mem<int> d_bodyBegIdx, d_bodyEndIdx;
  cuda_mem<unsigned long long> d_keys_inv;
  d_bodyBegIdx.alloc(numBodies);
  d_bodyEndIdx.alloc(numBodies);
  d_keys_inv.alloc(numBodies);
  groupTargets::mask_keys<<<nblock,nthread,(nthread+2)*sizeof(unsigned long long)>>>(numBodies, mask, d_keys, d_keys_inv, d_bodyBegIdx, d_bodyEndIdx);

  thrust::device_ptr<int> valuesBeg(d_bodyBegIdx.ptr);
  thrust::device_ptr<int> valuesEnd(d_bodyEndIdx.ptr);
  thrust::inclusive_scan_by_key(keys_beg,     keys_end,    valuesBeg, valuesBeg);

  thrust::device_ptr<unsigned long long> keys_inv_beg(d_keys_inv.ptr);
  thrust::device_ptr<unsigned long long> keys_inv_end(d_keys_inv.ptr + numBodies);
  thrust::inclusive_scan_by_key(keys_inv_beg, keys_inv_end, valuesEnd, valuesEnd);

  groupTargets::make_groups<<<nblock,nthread>>>(numBodies, NCRIT, d_bodyBegIdx, d_bodyEndIdx, d_targetCells);
#endif

  kernelSuccess("groupTargets");
  const double dt = get_time() - t0;
  fprintf(stdout,"Make groups          : %.7f s\n", dt);
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(&numTargets, HIP_SYMBOL(groupTargets::groupCounter), sizeof(int)));
}
#include "hip/hip_runtime.h"
#include "Treecode.h"
#include <algorithm>

#include "cuda_primitives.h"

#define CELL_LIST_MEM_PER_WARP (4096*32)
#define IF(x) (-(int)(x))

namespace computeForces {  
  texture<uint4,  1, hipReadModeElementType> texCellData;
  texture<float4, 1, hipReadModeElementType> texSourceCenter;
  texture<float4, 1, hipReadModeElementType> texCellMonopole;
  texture<float4, 1, hipReadModeElementType> texCellQuad0;
  texture<float2, 1, hipReadModeElementType> texCellQuad1;
  texture<float4, 1, hipReadModeElementType> texPtcl;

  static __device__ __forceinline__
  float6 make_float6(float xx, float yy, float zz, float xy, float xz, float yz) {
    float6 v;
    v.xx = xx;
    v.yy = yy;
    v.zz = zz;
    v.xy = xy;
    v.xz = xz;
    v.yz = yz;
    return v;
  }

  static __device__ __forceinline__
  int ringAddr(const int i) {
    return i & (CELL_LIST_MEM_PER_WARP - 1);
  }

  static __device__ __forceinline__
  bool applyMAC(const float4 sourceCenter,
                const float3 targetCenter,
                const float3 targetSize) {
    float3 dr = make_float3(fabsf(targetCenter.x - sourceCenter.x) - (targetSize.x),
                            fabsf(targetCenter.y - sourceCenter.y) - (targetSize.y),
                            fabsf(targetCenter.z - sourceCenter.z) - (targetSize.z));
    dr.x += fabsf(dr.x); dr.x *= 0.5f;
    dr.y += fabsf(dr.y); dr.y *= 0.5f;
    dr.z += fabsf(dr.z); dr.z *= 0.5f;
    const float ds2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
    return ds2 < fabsf(sourceCenter.w);
  }

  static __device__ __forceinline__
  float4 P2P(float4 acc,
             const float3 pos,
	     const float4 posj,
	     const float eps2) {
    const float3 dr = make_float3(posj.x - pos.x, posj.y - pos.y, posj.z - pos.z);
    const float r2     = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z + eps2;
    const float rinv   = rsqrtf(r2);
    const float rinv2  = rinv*rinv;
    const float mrinv  = posj.w * rinv;
    const float mrinv3 = mrinv * rinv2;
    acc.w -= mrinv;
    acc.x += mrinv3 * dr.x;
    acc.y += mrinv3 * dr.y;
    acc.z += mrinv3 * dr.z;
    return acc;
  }

  static __device__
  float4 M2P(float4 acc,
	     const float3 pos,
	     const float4 M0,
	     const float6 Q0,
	     float eps2) {
    const float3 dr = make_float3(pos.x - M0.x, pos.y - M0.y, pos.z - M0.z);
    const float  r2 = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + eps2;
    const float rinv  = rsqrtf(r2);
    const float rinv2 = rinv * rinv;
    const float mrinv  = M0.w * rinv;
    const float mrinv3 = rinv2 * mrinv;
    const float mrinv5 = rinv2 * mrinv3; 
    const float mrinv7 = rinv2 * mrinv5;
    const float  D0 =  mrinv;
    const float  D1 = -mrinv3;
    const float  D2 =  mrinv5 * 3.0f;
    const float  D3 = -mrinv7 * 15.0f;
    const float q11 = Q0.xx;
    const float q22 = Q0.yy;
    const float q33 = Q0.zz;
    const float q12 = Q0.xy;
    const float q13 = Q0.xz;
    const float q23 = Q0.yz;
    const float  q  = q11 + q22 + q33;
    const float3 qR = make_float3(
      q11 * dr.x + q12 * dr.y + q13 * dr.z,
      q12 * dr.x + q22 * dr.y + q23 * dr.z,
      q13 * dr.x + q23 * dr.y + q33 * dr.z);
    const float qRR = qR.x * dr.x + qR.y * dr.y + qR.z * dr.z;
    acc.w  -= D0 + 0.5f * (D1*q + D2 * qRR);
    const float C = D1 + 0.5f * (D2*q + D3 * qRR);
    acc.x  += C * dr.x + D2 * qR.x;
    acc.y  += C * dr.y + D2 * qR.y;
    acc.z  += C * dr.z + D2 * qR.z;
    return acc;
  }

  template<int NI, bool FULL>
  static __device__
  void approxAcc(float4 acc_i[NI],
		 const float3 pos_i[NI],
		 const int cellIdx,
		 const float eps2) {
    float4 M0, Q0;
    float2 Q1;
    if (FULL || cellIdx >= 0) {
      M0 = tex1Dfetch(texCellMonopole, cellIdx);
      Q0 = tex1Dfetch(texCellQuad0,    cellIdx);
      Q1 = tex1Dfetch(texCellQuad1,    cellIdx);
    } else {
      M0 = Q0 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
      Q1 = make_float2(0.0f, 0.0f);
    }
    for (int j=0; j<WARP_SIZE; j++) {
      const float4 jM0 = make_float4(__shfl(M0.x, j), __shfl(M0.y, j), __shfl(M0.z, j), __shfl(M0.w,j));
      const float6 jQ0 = make_float6(__shfl(Q0.x, j), __shfl(Q0.y, j), __shfl(Q0.z, j), __shfl(Q0.w,j),
				     __shfl(Q1.x, j), __shfl(Q1.y, j));
#pragma unroll
      for (int k=0; k<NI; k++)
	acc_i[k] = M2P(acc_i[k], pos_i[k], jM0, jQ0, eps2);
    }
  }

  template<int BLOCKDIM2, int NI>
  static __device__
  uint2 treewalk_warp(float4 acc_i[NI],
		      const float3 pos_i[NI],
		      const float3 targetCenter,
		      const float3 targetSize,
		      const float eps2,
		      const int2 top_cells,
		      int *shmem,
		      int *cellList) {
    const int laneIdx = threadIdx.x & (WARP_SIZE-1);

    uint2 counters = {0,0};

    volatile int *tmpList = shmem;

    int approxCellIdx, directPtclIdx;

    int directCounter = 0;
    int approxCounter = 0;

    for (int root_cell=top_cells.x; root_cell<top_cells.y; root_cell+=WARP_SIZE)
      if (root_cell + laneIdx < top_cells.y)
	cellList[ringAddr(root_cell - top_cells.x + laneIdx)] = root_cell + laneIdx;

    int nCells = top_cells.y - top_cells.x;

    int cellListBlock        = 0;
    int nextLevelCellCounter = 0;

    unsigned int cellListOffset = 0;

    /* process level with n_cells */
#if 1
    while (nCells > 0)
    {
      /* extract cell index from the current level cell list */
      const int cellListIdx = cellListBlock + laneIdx;
      const bool useCell    = cellListIdx < nCells;
      const int cellIdx     = cellList[ringAddr(cellListOffset + cellListIdx)];
      cellListBlock += min(WARP_SIZE, nCells - cellListBlock);

      /* read from gmem cell's info */
      const float4   sourceCenter = tex1Dfetch(texSourceCenter, cellIdx);
      const CellData cellData = tex1Dfetch(texCellData, cellIdx);

      const bool splitCell = applyMAC(sourceCenter, targetCenter, targetSize) ||
	(cellData.pend() - cellData.pbeg() < 3); /* force to open leaves with less than 3 particles */

      /**********************************************/
      /* split cells that satisfy opening condition */
      /**********************************************/

      const bool isNode = cellData.isNode();

      {
	const int firstChild = cellData.first();
	const int nChild= cellData.n();
	bool splitNode  = isNode && splitCell && useCell;

	/* use exclusive scan to compute scatter addresses for each of the child cells */
	const int2 childScatter = warpIntExclusiveScan(nChild & (-splitNode));

	/* make sure we still have available stack space */
	if (childScatter.y + nCells - cellListBlock > CELL_LIST_MEM_PER_WARP)
	  return make_uint2(0xFFFFFFFF,0xFFFFFFFF);

#if 1
	/* if so populate next level stack in gmem */
	if (splitNode)
	{
	  const int scatterIdx = cellListOffset + nCells + nextLevelCellCounter + childScatter.x;
	  for (int i = 0; i < nChild; i++)
	    cellList[ringAddr(scatterIdx + i)] = firstChild + i;
	}
#else  /* use scan operation to accomplish steps above, doesn't bring performance benefit */
	int nChildren  = childScatter.y;
	int nProcessed = 0;
	int2 scanVal   = {0,0};
	const int offset = cellListOffset + nCells + nextLevelCellCounter;
	while (nChildren > 0)
	{
	  tmpList[laneIdx] = 1;
	  if (splitNode && (childScatter.x - nProcessed < WARP_SIZE))
	  {
	    splitNode = false;
	    tmpList[childScatter.x - nProcessed] = -1-firstChild;
	  }
	  scanVal = inclusive_segscan_warp(tmpList[laneIdx], scanVal.y);
	  if (laneIdx < nChildren)
	    cellList[ringAddr(offset + nProcessed + laneIdx)] = scanVal.x;
	  nChildren  -= WARP_SIZE;
	  nProcessed += WARP_SIZE;
	}
#endif
	nextLevelCellCounter += childScatter.y;  /* increment nextLevelCounter by total # of children */
      }

#if 1
      {
	/***********************************/
	/******       APPROX          ******/
	/***********************************/

	/* see which thread's cell can be used for approximate force calculation */
	const bool approxCell    = !splitCell && useCell;
	const int2 approxScatter = warpBinExclusiveScan(approxCell);

	/* store index of the cell */
	const int scatterIdx = approxCounter + approxScatter.x;
	tmpList[laneIdx] = approxCellIdx;
	if (approxCell && scatterIdx < WARP_SIZE)
	  tmpList[scatterIdx] = cellIdx;

	approxCounter += approxScatter.y;

	/* compute approximate forces */
	if (approxCounter >= WARP_SIZE)
	{
	  /* evalute cells stored in shmem */
	  approxAcc<NI,true>(acc_i, pos_i, tmpList[laneIdx], eps2);

	  approxCounter -= WARP_SIZE;
	  const int scatterIdx = approxCounter + approxScatter.x - approxScatter.y;
	  if (approxCell && scatterIdx >= 0)
	    tmpList[scatterIdx] = cellIdx;
	  counters.x += WARP_SIZE;
	}
	approxCellIdx = tmpList[laneIdx];
      }
#endif

#if 1
      {
	/***********************************/
	/******       DIRECT          ******/
	/***********************************/

	const bool isLeaf = !isNode;
	bool isDirect = splitCell && isLeaf && useCell;

	const int firstBody = cellData.pbeg();
	const int     nBody = cellData.pend() - cellData.pbeg();

	const int2 childScatter = warpIntExclusiveScan(nBody & (-isDirect));
	int nParticle  = childScatter.y;
	int nProcessed = 0;
	int2 scanVal   = {0,0};

	/* conduct segmented scan for all leaves that need to be expanded */
	while (nParticle > 0)
	{
	  tmpList[laneIdx] = 1;
	  if (isDirect && (childScatter.x - nProcessed < WARP_SIZE))
	  {
	    isDirect = false;
	    tmpList[childScatter.x - nProcessed] = -1-firstBody;
	  }
	  scanVal = inclusive_segscan_warp(tmpList[laneIdx], scanVal.y);
	  const int  ptclIdx = scanVal.x;

	  if (nParticle >= WARP_SIZE)
	  {
	    const float4 M0 = tex1Dfetch(texPtcl, ptclIdx);
	    for (int j=0; j<WARP_SIZE; j++) {
	      const float4 pos_j = make_float4(__shfl(M0.x, j), __shfl(M0.y, j), __shfl(M0.z, j), __shfl(M0.w,j));
#pragma unroll
	      for (int k=0; k<NI; k++)
		acc_i[k] = P2P(acc_i[k], pos_i[k], pos_j, eps2);
	    }
	    nParticle  -= WARP_SIZE;
	    nProcessed += WARP_SIZE;
	    counters.y += WARP_SIZE;
	  }
	  else 
	  {
	    const int scatterIdx = directCounter + laneIdx;
	    tmpList[laneIdx] = directPtclIdx;
	    if (scatterIdx < WARP_SIZE)
	      tmpList[scatterIdx] = ptclIdx;

	    directCounter += nParticle;

	    if (directCounter >= WARP_SIZE)
	    {
	      /* evalute cells stored in shmem */
	      const float4 M0 = tex1Dfetch(texPtcl, tmpList[laneIdx]);
	      for (int j=0; j<WARP_SIZE; j++) {
		const float4 pos_j = make_float4(__shfl(M0.x, j), __shfl(M0.y, j), __shfl(M0.z, j), __shfl(M0.w,j));
#pragma unroll
		for (int k=0; k<NI; k++)
		  acc_i[k] = P2P(acc_i[k], pos_i[k], pos_j, eps2);
	      }
	      directCounter -= WARP_SIZE;
	      const int scatterIdx = directCounter + laneIdx - nParticle;
	      if (scatterIdx >= 0)
		tmpList[scatterIdx] = ptclIdx;
	      counters.y += WARP_SIZE;
	    }
	    directPtclIdx = tmpList[laneIdx];

	    nParticle = 0;
	  }
	}
      }
#endif

      /* if the current level is processed, schedule the next level */
      if (cellListBlock >= nCells)
      {
	cellListOffset += nCells;
	nCells = nextLevelCellCounter;
	cellListBlock = nextLevelCellCounter = 0;
      }

    }  /* level completed */
#endif

#if 1
    if (approxCounter > 0)
    {
      approxAcc<NI,false>(acc_i, pos_i, laneIdx < approxCounter ? approxCellIdx : -1, eps2);
      counters.x += approxCounter;
      approxCounter = 0;
    }
#endif

#if 1
    if (directCounter > 0)
    {
      const int ptclIdx = laneIdx < directCounter ? directPtclIdx : -1;
      const float4 M0 = ptclIdx >= 0 ? tex1Dfetch(texPtcl, ptclIdx) : make_float4(0.0f, 0.0f, 0.0f, 0.0f);
      for (int j=0; j<WARP_SIZE; j++) {
	const float4 pos_j = make_float4(__shfl(M0.x, j), __shfl(M0.y, j), __shfl(M0.z, j), __shfl(M0.w,j));
#pragma unroll
	for (int k=0; k<NI; k++)
	  acc_i[k] = P2P(acc_i[k], pos_i[k], pos_j, eps2);
      }
      counters.y += directCounter;
      directCounter = 0;
    }
#endif

    return counters;
  }

  __device__ unsigned int retired_groupCount = 0;

  __device__ unsigned long long g_direct_sum = 0;
  __device__ unsigned int       g_direct_max = 0;

  __device__ unsigned long long g_approx_sum = 0;
  __device__ unsigned int       g_approx_max = 0;

  template<int NTHREAD2, int NI>
    __launch_bounds__(1<<NTHREAD2, 1024/(1<<NTHREAD2))
    static __global__ 
    void treewalk(
        const int nGroups,
        const int2 *groupList,
        const float eps2,
        const int start_level,
        const int2 *level_begIdx,
        const float4 *pos,
        float4 *acc,
        int    *gmem_pool)
    {
      const int NTHREAD = 1<<NTHREAD2;
      const int shMemSize = NTHREAD;
      __shared__ int shmem_pool[shMemSize];

      const int laneIdx = threadIdx.x & (WARP_SIZE-1);
      const int warpIdx = threadIdx.x >> WARP_SIZE2;

      const int NWARP2 = NTHREAD2 - WARP_SIZE2;
      const int sh_offs = (shMemSize >> NWARP2) * warpIdx;
      int *shmem = shmem_pool + sh_offs;
      int *gmem  =  gmem_pool + CELL_LIST_MEM_PER_WARP*((blockIdx.x<<NWARP2) + warpIdx);

      int2 top_cells = level_begIdx[start_level];
      top_cells.y++;

      while (1) {
        int groupIdx = 0;
        if (laneIdx == 0)
          groupIdx = atomicAdd(&retired_groupCount, 1);
        groupIdx = __shfl(groupIdx, 0, WARP_SIZE);

        if (groupIdx >= nGroups) 
          return;

        const int2 group = groupList[groupIdx];
        const int begin = group.x;
        const int end   = group.x+group.y;

        float3 pos_i[NI];
#pragma unroll
        for (int i=0; i<NI; i++) {
          const float4 ptcl = pos[min(begin+i*WARP_SIZE+laneIdx,end-1)];
          pos_i[i] = make_float3(ptcl.x, ptcl.y, ptcl.z);
        }
        float3 rmin = pos_i[0];
        float3 rmax = rmin; 
#pragma unroll
        for (int i = 0; i < NI; i++) 
          addBoxSize(rmin, rmax, pos_i[i]);
        rmin.x = __shfl(rmin.x,0);
        rmin.y = __shfl(rmin.y,0);
        rmin.z = __shfl(rmin.z,0);
        rmax.x = __shfl(rmax.x,0);
        rmax.y = __shfl(rmax.y,0);
        rmax.z = __shfl(rmax.z,0);

        const float half = 0.5f;
        const float3 targetCenter = {half*(rmax.x+rmin.x), half*(rmax.y+rmin.y), half*(rmax.z+rmin.z)};
        const float3 hvec = {half*(rmax.x-rmin.x), half*(rmax.y-rmin.y), half*(rmax.z-rmin.z)};

        float4 acc_i[NI] = {0.0f, 0.0f, 0.0f, 0.0f};

        uint2 counters = treewalk_warp<NTHREAD2,NI>
          (acc_i, pos_i, targetCenter, hvec, eps2, top_cells, shmem, gmem);

        assert(!(counters.x == 0xFFFFFFFF && counters.y == 0xFFFFFFFF));

        const int pidx = begin + laneIdx;

	int direct_max = counters.y;
	int direct_sum = 0;
	int approx_max = counters.x;
	int approx_sum = 0;

#pragma unroll
	for (int i = 0; i < NI; i++)
	  if (i*WARP_SIZE + pidx < end)
	  {
	    approx_sum += counters.x;
	    direct_sum += counters.y;
	  }

#pragma unroll
	for (int i = WARP_SIZE2-1; i >= 0; i--)
	{
	  direct_max  = max(direct_max, __shfl_xor(direct_max, 1<<i));
	  direct_sum += __shfl_xor(direct_sum, 1<<i);
	  approx_max  = max(approx_max, __shfl_xor(approx_max, 1<<i));
	  approx_sum += __shfl_xor(approx_sum, 1<<i);
	}

	if (laneIdx == 0)
	{
	  atomicMax(&g_direct_max,                     direct_max);
	  atomicAdd(&g_direct_sum, (unsigned long long)direct_sum);
	  atomicMax(&g_approx_max,                     approx_max);
	  atomicAdd(&g_approx_sum, (unsigned long long)approx_sum);
	}

#pragma unroll
        for (int i=0; i<NI; i++)
          if (pidx + i * WARP_SIZE < end)
            acc[i*WARP_SIZE + pidx] = acc_i[i];
      }
    }

  static __global__
  void direct(const int numSource,
              const float eps2,
	      float4 *acc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockIdx.x * numSource / gridDim.x;
    float pots, axs, ays ,azs;
    float potc, axc, ayc ,azc;
    float4 si = tex1Dfetch(texPtcl, threadIdx.x);
    __shared__ float4 s[512];
    for ( int jb=0; jb<numSource/blockDim.x/gridDim.x; jb++ ) {
      __syncthreads();
      s[threadIdx.x] = tex1Dfetch(texPtcl, offset+jb*blockDim.x+threadIdx.x);
      __syncthreads();
      for( int j=0; j<blockDim.x; j++ ) {
	float dx = s[j].x - si.x;
	float dy = s[j].y - si.y;
	float dz = s[j].z - si.z;
	float R2 = dx * dx + dy * dy + dz * dz + eps2;
	float invR = rsqrtf(R2);
        float y = - s[j].w * invR - potc;
        float t = pots + y;
        potc = (t - pots) - y;
        pots = t;
	float invR3 = invR * invR * invR * s[j].w;
        y = dx * invR3 - axc;
        t = axs + y;
        axc = (t - axs) - y;
        axs = t;
        y = dy * invR3 - ayc;
        t = ays + y;
        ayc = (t - ays) - y;
        ays = t;
        y = dz * invR3 - azc;
        t = azs + y;
        azc = (t - azs) - y;
        azs = t;
      }
    }
    acc[i].x = axs + axc;
    acc[i].y = ays + ayc;
    acc[i].z = azs + azc;
    acc[i].w = pots + potc;
  }
}

float4 Treecode::computeForces() {
  bindTexture(computeForces::texCellData,(uint4*)d_cellDataList.ptr, nCells);
  bindTexture(computeForces::texSourceCenter,    d_sourceCenter.ptr, nCells);
  bindTexture(computeForces::texCellMonopole,    d_cellMonopole.ptr, nCells);
  bindTexture(computeForces::texCellQuad0,       d_cellQuad0.ptr,    nCells);
  bindTexture(computeForces::texCellQuad1,       d_cellQuad1.ptr,    nCells);
  bindTexture(computeForces::texPtcl,            d_ptclPos.ptr,      nPtcl);

  const int NTHREAD2 = 7;
  const int NTHREAD  = 1<<NTHREAD2;
  cuda_mem<int> d_gmem_pool;

  const int nblock = 8*13;
  d_gmem_pool.alloc(CELL_LIST_MEM_PER_WARP*nblock*(NTHREAD/WARP_SIZE));

  const int starting_level = 1;
  int value = 0;
  hipDeviceSynchronize();
  const double t0 = get_time();
  unsigned long long lzero = 0;
  unsigned int       uzero = 0;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(computeForces::retired_groupCount), &value, sizeof(int)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(computeForces::g_direct_sum), &lzero, sizeof(unsigned long long)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(computeForces::g_direct_max), &uzero, sizeof(unsigned int)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(computeForces::g_approx_sum), &lzero, sizeof(unsigned long long)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(computeForces::g_approx_max), &uzero, sizeof(unsigned int)));

  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&computeForces::treewalk<NTHREAD2),2>, hipFuncCachePreferL1));
  computeForces::treewalk<NTHREAD2,2><<<nblock,NTHREAD>>>(
    nGroups, d_groupList, eps2, starting_level, d_level_begIdx,
    d_ptclPos_tmp, d_ptclAcc,
    d_gmem_pool);
  kernelSuccess("treewalk");
  const double dt = get_time() - t0;

  float4 interactions = {0.0, 0.0, 0.0, 0.0};
  unsigned long long direct_sum, approx_sum;
  unsigned int direct_max, approx_max;
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(&direct_sum, HIP_SYMBOL(computeForces::g_direct_sum), sizeof(unsigned long long)));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(&direct_max, HIP_SYMBOL(computeForces::g_direct_max), sizeof(unsigned int)));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(&approx_sum, HIP_SYMBOL(computeForces::g_approx_sum), sizeof(unsigned long long)));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(&approx_max, HIP_SYMBOL(computeForces::g_approx_max), sizeof(unsigned int)));
  interactions.x = direct_sum*1.0/nPtcl;
  interactions.y = direct_max;
  interactions.z = approx_sum*1.0/nPtcl;
  interactions.w = approx_max;

  float flops = (interactions.x*20 + interactions.z*64)*nPtcl/dt/1e12;
  fprintf(stdout,"Traverse             : %.7f s (%.7f TFlops)\n",dt,flops);

  unbindTexture(computeForces::texPtcl);
  unbindTexture(computeForces::texCellQuad1);
  unbindTexture(computeForces::texCellQuad0);
  unbindTexture(computeForces::texCellMonopole);
  unbindTexture(computeForces::texSourceCenter);
  unbindTexture(computeForces::texCellData);

  return interactions;
}

void Treecode::computeDirect(const int numTarget, const int numBlock)
{
  bindTexture(computeForces::texPtcl,d_ptclPos_tmp.ptr,nPtcl);
  computeForces::direct<<<numBlock,numTarget>>>(nPtcl, eps2, d_ptclAcc2);
  unbindTexture(computeForces::texPtcl);
  hipDeviceSynchronize();
}

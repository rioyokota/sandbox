#include "hip/hip_runtime.h"
#include "Treecode.h"
#include "cuda_primitives.h"

#include "buildtree.h"
#include "upwardpass.h"
#include "grouptargets.h"
#include "traversal.h"

int main(int argc, char * argv[])
{
  const int numBodies = 16777216;
  const int seed = 19810614;
  const float eps   = 0.05;
  const float theta = 0.75;
  const int ncrit = 64;
  const int nleaf = 64;

  fprintf(stdout,"--- FMM Parameters ---------------\n");
  fprintf(stdout,"numBodies            : %d\n",numBodies);
  fprintf(stdout,"P                    : %d\n",3);
  fprintf(stdout,"theta                : %f\n",theta);
  fprintf(stdout,"ncrit                : %d\n",ncrit);
  fprintf(stdout,"nleaf                : %d\n",nleaf);
  const Plummer data(numBodies, seed);

  host_mem<float4> h_bodyPos;
  h_bodyPos.alloc(numBodies);
  
  for (int i = 0; i < numBodies; i++) {
    float4 bodyPos;
    bodyPos.x    = data.pos[i].x;
    bodyPos.y    = data.pos[i].y;
    bodyPos.z    = data.pos[i].z;
    bodyPos.w    = data.mass[i];
    h_bodyPos[i] = bodyPos;
  }
  cuda_mem<float4> d_bodyPos;
  cuda_mem<float4> d_bodyPos2;
  cuda_mem<float4> d_bodyAcc;
  cuda_mem<float4> d_bodyAcc2;
  d_bodyPos.alloc(numBodies);
  d_bodyPos2.alloc(numBodies);
  d_bodyAcc.alloc(numBodies);
  d_bodyAcc2.alloc(numBodies);
  d_bodyPos.h2d(h_bodyPos);
  d_bodyAcc.h2d(h_bodyPos);

  cuda_mem<CellData> d_sourceCells;
  cuda_mem<int2> d_targetCells;
  cuda_mem<float4> d_sourceCenter;
  cuda_mem<float4> d_Monopole;
  cuda_mem<float4> d_Quadrupole0;
  cuda_mem<float2> d_Quadrupole1;
  cuda_mem<float4> d_domain;
  cuda_mem<int2> d_levelRange;
  d_domain.alloc(1);
  d_levelRange.alloc(32);
  d_sourceCells.alloc(numBodies);
  d_targetCells.alloc(numBodies);

  fprintf(stdout,"--- FMM Profiling ----------------\n");
  double t0 = get_time();
  int2 numLS = treeBuild::buildTree(numBodies, d_bodyPos, d_bodyPos2, d_bodyAcc, d_domain, d_levelRange, d_sourceCells, nleaf); // pass nleaf, accepted 16, 24, 32, 48, 64
  int numLevels = numLS.x;
  int numSources = numLS.y;
  d_sourceCenter.alloc(numSources);
  d_Monopole.alloc(numSources);
  d_Quadrupole0.alloc(numSources);
  d_Quadrupole1.alloc(numSources);
  multipoles::computeMultipoles(numBodies, numSources, theta, d_bodyPos, d_sourceCells, d_sourceCenter, d_Monopole, d_Quadrupole0, d_Quadrupole1);
  int numTargets = groupTargets::groupTargets(numBodies, d_bodyPos, d_bodyPos2, d_domain, d_targetCells, 5, ncrit);
  const float4 interactions = computeForces::computeForces(numBodies, numTargets, numSources, eps, d_bodyPos, d_bodyPos2, d_bodyAcc,
						 d_sourceCells, d_targetCells, d_sourceCenter, d_Monopole, d_Quadrupole0, d_Quadrupole1, d_levelRange);
  double dt = get_time() - t0;
  float flops = (interactions.x * 20 + interactions.z * 64) * numBodies / dt / 1e12;
  fprintf(stdout,"--- Total runtime ----------------\n");
  fprintf(stdout,"Total FMM            : %.7f s (%.7f TFlops)\n",dt,flops);
  const int numTarget = 512; // Number of threads per block will be set to this value
  const int numBlock = 128;
  t0 = get_time();
  computeForces::computeDirect(numBodies, numTarget, numBlock, eps, d_bodyPos2, d_bodyAcc2);
  dt = get_time() - t0;
  flops = 20.*numTarget*numBodies/dt/1e12;
  fprintf(stdout,"Total Direct         : %.7f s (%.7f TFlops)\n",dt,flops);
  host_mem<float4> h_bodyAcc, h_bodyAcc2;
  h_bodyAcc.alloc(numBodies);
  h_bodyAcc2.alloc(numBodies);
  d_bodyAcc.d2h(h_bodyAcc);
  d_bodyAcc2.d2h(h_bodyAcc2);

  for (int i=0; i<numTarget; i++) {
    float4 bodyAcc = h_bodyAcc2[i];
    for (int j=1; j<numBlock; j++) {
      bodyAcc.x += h_bodyAcc2[i+numTarget*j].x;
      bodyAcc.y += h_bodyAcc2[i+numTarget*j].y;
      bodyAcc.z += h_bodyAcc2[i+numTarget*j].z;
      bodyAcc.w += h_bodyAcc2[i+numTarget*j].w;
    }
    h_bodyAcc2[i] = bodyAcc;
  }

  double diffp = 0, diffa = 0;
  double normp = 0, norma = 0;
  for (int i=0; i<numTarget; i++) {
    diffp += (h_bodyAcc[i].w - h_bodyAcc2[i].w) * (h_bodyAcc[i].w - h_bodyAcc2[i].w);
    diffa += (h_bodyAcc[i].x - h_bodyAcc2[i].x) * (h_bodyAcc[i].x - h_bodyAcc2[i].x)
      + (h_bodyAcc[i].y - h_bodyAcc2[i].y) * (h_bodyAcc[i].y - h_bodyAcc2[i].y)
      + (h_bodyAcc[i].z - h_bodyAcc2[i].z) * (h_bodyAcc[i].z - h_bodyAcc2[i].z);
    normp += h_bodyAcc2[i].w * h_bodyAcc2[i].w;
    norma += h_bodyAcc2[i].x * h_bodyAcc2[i].x
      + h_bodyAcc2[i].y * h_bodyAcc2[i].y
      + h_bodyAcc2[i].z * h_bodyAcc2[i].z;
  }
  fprintf(stdout,"--- FMM vs. direct ---------------\n");
  fprintf(stdout,"Rel. L2 Error (pot)  : %.7e\n",sqrt(diffp/normp));
  fprintf(stdout,"Rel. L2 Error (acc)  : %.7e\n",sqrt(diffa/norma));
  fprintf(stdout,"--- Tree stats -------------------\n");
  fprintf(stdout,"Bodies               : %d\n",numBodies);
  fprintf(stdout,"Cells                : %d\n",numSources);
  fprintf(stdout,"Tree depth           : %d\n",numLevels);
  fprintf(stdout,"--- Traversal stats --------------\n");
  fprintf(stdout,"P2P mean list length : %g (max %g)\n", interactions.x, interactions.y);
  fprintf(stdout,"M2P mean list length : %g (max %g)\n", interactions.z, interactions.w);
  return 0;
}
#include "hip/hip_runtime.h"
#include "types.h"
#include "cuda_primitives.h"
#include "buildtree.h"
#include "upwardpass.h"
#include "grouptargets.h"
#include "traversal.h"

int main(int argc, char * argv[]) {
  const int numBodies = (1 << 24) - 1;
  const float eps = 0.05;
  const float theta = 0.75;
  const int ncrit = 64;

  fprintf(stdout,"--- FMM Parameters ---------------\n");
  fprintf(stdout,"numBodies            : %d\n",numBodies);
  fprintf(stdout,"P                    : %d\n",3);
  fprintf(stdout,"theta                : %f\n",theta);
  fprintf(stdout,"ncrit                : %d\n",ncrit);
  const Plummer data(numBodies);

  cudaVec<float4> bodyPos(numBodies);
  cudaVec<float4> bodyPos2(numBodies);
  cudaVec<float4> bodyAcc(numBodies);
  cudaVec<float4> bodyAcc2(numBodies);
  for (int i=0; i<numBodies; i++) {
    bodyPos[i].x = data.pos[i].x;
    bodyPos[i].y = data.pos[i].y;
    bodyPos[i].z = data.pos[i].z;
    bodyPos[i].w = data.pos[i].w;
  }
  bodyPos.h2d();
  bodyAcc.h2d();

  cuda_mem<int2> d_targetRange;
  cuda_mem<CellData> d_sourceCells;
  cuda_mem<float4> d_sourceCenter;
  cuda_mem<float4> d_Monopole;
  cuda_mem<float4> d_Quadrupole0;
  cuda_mem<float2> d_Quadrupole1;
  cuda_mem<int2> d_levelRange;
  d_levelRange.alloc(32);
  d_targetRange.alloc(numBodies);
  d_sourceCells.alloc(numBodies);

  fprintf(stdout,"--- FMM Profiling ----------------\n");
  double t0 = get_time();
  Build build;
  float4 domain;
  int2 numLS = build.tree<ncrit>(numBodies, bodyPos.devc(), bodyPos2.devc(), domain, d_levelRange, d_sourceCells);
  int numLevels = numLS.x;
  int numSources = numLS.y;
  d_sourceCenter.alloc(numSources);
  d_Monopole.alloc(numSources);
  d_Quadrupole0.alloc(numSources);
  d_Quadrupole1.alloc(numSources);
  Group group;
  int numTargets = group.targets(numBodies, bodyPos.devc(), bodyPos2.devc(), domain, d_targetRange, 5);
  Pass pass;
  pass.upward(numBodies, numSources, theta, bodyPos.devc(), d_sourceCells, d_sourceCenter, d_Monopole, d_Quadrupole0, d_Quadrupole1);
  Traversal traversal;
  const float4 interactions = traversal.approx(numBodies, numTargets, numSources, eps,
					       bodyPos.devc(), bodyPos2.devc(), bodyAcc.devc(),
					       d_targetRange, d_sourceCells, d_sourceCenter,
					       d_Monopole, d_Quadrupole0, d_Quadrupole1, d_levelRange);
  double dt = get_time() - t0;
  float flops = (interactions.x * 20 + interactions.z * 64) * numBodies / dt / 1e12;
  fprintf(stdout,"--- Total runtime ----------------\n");
  fprintf(stdout,"Total FMM            : %.7f s (%.7f TFlops)\n",dt,flops);
  const int numTarget = 512; // Number of threads per block will be set to this value
  const int numBlock = 128;
  t0 = get_time();
  traversal.direct(numBodies, numTarget, numBlock, eps, bodyPos2.devc(), bodyAcc2.devc());
  dt = get_time() - t0;
  flops = 35.*numTarget*numBodies/dt/1e12;
  fprintf(stdout,"Total Direct         : %.7f s (%.7f TFlops)\n",dt,flops);
  bodyAcc.d2h();
  bodyAcc2.d2h();

  for (int i=0; i<numTarget; i++) {
    float4 bodyAcc = bodyAcc2[i];
    for (int j=1; j<numBlock; j++) {
      bodyAcc.x += bodyAcc2[i+numTarget*j].x;
      bodyAcc.y += bodyAcc2[i+numTarget*j].y;
      bodyAcc.z += bodyAcc2[i+numTarget*j].z;
      bodyAcc.w += bodyAcc2[i+numTarget*j].w;
    }
    bodyAcc2[i] = bodyAcc;
  }

  double diffp = 0, diffa = 0;
  double normp = 0, norma = 0;
  for (int i=0; i<numTarget; i++) {
    diffp += (bodyAcc[i].w - bodyAcc2[i].w) * (bodyAcc[i].w - bodyAcc2[i].w);
    diffa += (bodyAcc[i].x - bodyAcc2[i].x) * (bodyAcc[i].x - bodyAcc2[i].x)
      + (bodyAcc[i].y - bodyAcc2[i].y) * (bodyAcc[i].y - bodyAcc2[i].y)
      + (bodyAcc[i].z - bodyAcc2[i].z) * (bodyAcc[i].z - bodyAcc2[i].z);
    normp += bodyAcc2[i].w * bodyAcc2[i].w;
    norma += bodyAcc2[i].x * bodyAcc2[i].x
      + bodyAcc2[i].y * bodyAcc2[i].y
      + bodyAcc2[i].z * bodyAcc2[i].z;
  }
  fprintf(stdout,"--- FMM vs. direct ---------------\n");
  fprintf(stdout,"Rel. L2 Error (pot)  : %.7e\n",sqrt(diffp/normp));
  fprintf(stdout,"Rel. L2 Error (acc)  : %.7e\n",sqrt(diffa/norma));
  fprintf(stdout,"--- Tree stats -------------------\n");
  fprintf(stdout,"Bodies               : %d\n",numBodies);
  fprintf(stdout,"Cells                : %d\n",numSources);
  fprintf(stdout,"Tree depth           : %d\n",numLevels);
  fprintf(stdout,"--- Traversal stats --------------\n");
  fprintf(stdout,"P2P mean list length : %g (max %g)\n", interactions.x, interactions.y);
  fprintf(stdout,"M2P mean list length : %g (max %g)\n", interactions.z, interactions.w);
  return 0;
}
#include "hip/hip_runtime.h"
#include "Treecode.h"

int main(int argc, char * argv[])
{
  typedef Treecode Tree;

  const int numBodies = 16777216;
  const int seed = 19810614;
  const float eps   = 0.05;
  const float THETA = 0.75;
  const int NCRIT = 64;
  const int NLEAF = 64;
  Tree tree(eps, THETA);

  fprintf(stdout,"--- FMM Parameters ---------------\n");
  fprintf(stdout,"numBodies            : %d\n",numBodies);
  fprintf(stdout,"P                    : %d\n",3);
  fprintf(stdout,"THETA                : %f\n",THETA);
  fprintf(stdout,"NCRIT                : %d\n",NCRIT);
  fprintf(stdout,"NLEAF                : %d\n",NLEAF);
  const Plummer data(numBodies, seed);

  host_mem<float4> h_bodyPos;
  h_bodyPos.alloc(numBodies);
  
  tree.alloc(numBodies);
  for (int i = 0; i < numBodies; i++) {
    float4 bodyPos;
    bodyPos.x    = data.pos[i].x;
    bodyPos.y    = data.pos[i].y;
    bodyPos.z    = data.pos[i].z;
    bodyPos.w    = data.mass[i];
    h_bodyPos[i] = bodyPos;
  }
  tree.d_bodyPos.h2d(h_bodyPos);
  tree.d_bodyAcc2.h2d(h_bodyPos);

  cuda_mem<float4> d_domain;
  cuda_mem<int2> d_levelRange;
  d_domain.alloc(1);
  d_levelRange.alloc(32);

  fprintf(stdout,"--- FMM Profiling ----------------\n");
  double t0 = get_time();
  tree.buildTree(d_domain, d_levelRange, NLEAF); // pass NLEAF, accepted 16, 24, 32, 48, 64
  tree.computeMultipoles();
  tree.groupTargets(d_domain, 5, NCRIT);
  const float4 interactions = tree.computeForces(d_levelRange);
  double dt = get_time() - t0;
  float flops = (interactions.x * 20 + interactions.z * 64) * tree.getNumBody() / dt / 1e12;
  fprintf(stdout,"--- Total runtime ----------------\n");
  fprintf(stdout,"Total FMM            : %.7f s (%.7f TFlops)\n",dt,flops);
  const int numTarget = 512; // Number of threads per block will be set to this value
  const int numBlock = 128;
  t0 = get_time();
  tree.computeDirect(numTarget,numBlock);
  dt = get_time() - t0;
  flops = 20.*numTarget*numBodies/dt/1e12;
  fprintf(stdout,"Total Direct         : %.7f s (%.7f TFlops)\n",dt,flops);
  host_mem<float4> h_bodyAcc, h_bodyAcc2;
  h_bodyAcc.alloc(numBodies);
  h_bodyAcc2.alloc(numBodies);
  tree.d_bodyAcc.d2h(h_bodyAcc);
  tree.d_bodyAcc2.d2h(h_bodyAcc2);

  for (int i=0; i<numTarget; i++) {
    float4 bodyAcc = h_bodyAcc2[i];
    for (int j=1; j<numBlock; j++) {
      bodyAcc.x += h_bodyAcc2[i+numTarget*j].x;
      bodyAcc.y += h_bodyAcc2[i+numTarget*j].y;
      bodyAcc.z += h_bodyAcc2[i+numTarget*j].z;
      bodyAcc.w += h_bodyAcc2[i+numTarget*j].w;
    }
    h_bodyAcc2[i] = bodyAcc;
  }

  double diffp = 0, diffa = 0;
  double normp = 0, norma = 0;
  for (int i=0; i<numTarget; i++) {
    diffp += (h_bodyAcc[i].w - h_bodyAcc2[i].w) * (h_bodyAcc[i].w - h_bodyAcc2[i].w);
    diffa += (h_bodyAcc[i].x - h_bodyAcc2[i].x) * (h_bodyAcc[i].x - h_bodyAcc2[i].x)
      + (h_bodyAcc[i].y - h_bodyAcc2[i].y) * (h_bodyAcc[i].y - h_bodyAcc2[i].y)
      + (h_bodyAcc[i].z - h_bodyAcc2[i].z) * (h_bodyAcc[i].z - h_bodyAcc2[i].z);
    normp += h_bodyAcc2[i].w * h_bodyAcc2[i].w;
    norma += h_bodyAcc2[i].x * h_bodyAcc2[i].x
      + h_bodyAcc2[i].y * h_bodyAcc2[i].y
      + h_bodyAcc2[i].z * h_bodyAcc2[i].z;
  }
  fprintf(stdout,"--- FMM vs. direct ---------------\n");
  fprintf(stdout,"Rel. L2 Error (pot)  : %.7e\n",sqrt(diffp/normp));
  fprintf(stdout,"Rel. L2 Error (acc)  : %.7e\n",sqrt(diffa/norma));
  fprintf(stdout,"--- Tree stats -------------------\n");
  fprintf(stdout,"Bodies               : %d\n",tree.getNumBody());
  fprintf(stdout,"Cells                : %d\n",tree.getNumSources());
  fprintf(stdout,"Tree depth           : %d\n",tree.getNumLevels());
  fprintf(stdout,"--- Traversal stats --------------\n");
  fprintf(stdout,"P2P mean list length : %g (max %g)\n", interactions.x, interactions.y);
  fprintf(stdout,"M2P mean list length : %g (max %g)\n", interactions.z, interactions.w);
  return 0;
}
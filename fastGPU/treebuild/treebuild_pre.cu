#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <vector>
#include "rtc.h"
#include "plummer.h"
#include "cudamem.h"

#define __out 

#define WARP_SIZE2 5
#define WARP_SIZE  32

__device__ unsigned int nnodes = 0;
__device__ unsigned int nleaves = 0;
__device__ unsigned int nlevels = 0;
__device__ unsigned int nbodies_leaf = 0;
__device__ unsigned int ncells = 0;


__device__   int *memPool;
__constant__ int d_node_max;
__constant__ int d_cell_max;
__device__ unsigned long long io_words;

template<int N, typename T> struct vec;
template<> struct vec<4,float>  { typedef float4  type; };
template<> struct vec<4,double> { typedef double4 type; };

template<typename T> struct int_type;
template<> struct int_type<float>  { typedef int       type; };
template<> struct int_type<double> { typedef long long type; };

template<typename T> 
struct Particle4
{
  typedef typename int_type<T>::type intx;
  private:
  union
  {
    typename vec<4,T>::type packed_data;
    struct {double _x,_y,_z; intx _id;};
  };
  public:

  __host__ __device__ T x   ()  const { return packed_data.x;}
  __host__ __device__ T y   ()  const { return packed_data.y;}
  __host__ __device__ T z   ()  const { return packed_data.z;}
  __host__ __device__ T mass()  const { return packed_data.w;}
  __host__ __device__ intx id() const { return _id; }

  __host__ __device__ T& x    () { return packed_data.x;}
  __host__ __device__ T& y    () { return packed_data.y;}
  __host__ __device__ T& z    () { return packed_data.z;}
  __host__ __device__ T& mass () { return packed_data.w;}
  __host__ __device__ intx& id() { return _id; }
};

template<typename T>
struct Position
{
  T x, y, z;
  __host__ __device__ Position() {}
  __host__ __device__ Position(const T _x) : x(_x), y(_x), z(_x) {}

  __host__ __device__ Position(const T _x, const T _y, const T _z) : x(_x), y(_y), z(_z) {}
  static __host__ __device__ Position min(const Position &lhs, const Position &rhs) 
  {
    return Position( 
        fmin(lhs.x, rhs.x),
        fmin(lhs.y, rhs.y),
        fmin(lhs.z, rhs.z));
  }
  static __host__ __device__ Position max(const Position &lhs, const Position &rhs) 
  {
    return Position( 
        fmax(lhs.x, rhs.x),
        fmax(lhs.y, rhs.y),
        fmax(lhs.z, rhs.z));
  }
  __forceinline__ __device__ void shfl(const Position &p, const int i);
};

  template<typename T>
static __forceinline__ __device__ T myshfl(const T var0, T var, const int srcLane)
{
  var = __shfl(var, srcLane, WARP_SIZE);
  return srcLane < WARP_SIZE ? var : var0;
}

  template<>
__forceinline__ __device__ void Position<float>::shfl(const Position<float> &p, const int i)
{
  x = myshfl(x, p.x, i);
  y = myshfl(y, p.y, i);
  z = myshfl(z, p.z, i);
}


  template<typename T>
static __forceinline__ __device__ Position<T> get_volatile(const volatile Position<T>  &v)
{
  return Position<T>(v.x, v.y, v.z);
};

template<typename T>
struct BoundingBox
{
  Position<T> min, max;
  __device__ BoundingBox() {}
  __device__ BoundingBox(const Position<T> &_min, const Position<T> &_max) : min(_min), max(_max) {}
  __device__ Position<T> centre() const {return Position<T>(T(0.5)*(max.x + min.x), T(0.5)*(max.y + min.y), T(0.5)*(max.z + min.z)); }
  __device__ Position<T>  hsize() const {return Position<T>(T(0.5)*(max.x - min.x), T(0.5)*(max.y - min.y), T(0.5)*(max.z - min.z)); }
};

template<typename T>
struct __align__(4) ParticleLight
{
  Position<T> pos;
  float     idFlt;
  __host__ __device__ ParticleLight() {}
  __host__ ParticleLight(const Position<T> &_pos, const int _id) : pos(_pos), idFlt(*(float*)&_id) {}
  __device__ int d_id() const {return __float_as_int(idFlt); }
  __device__ void shfl(const ParticleLight &p, const int i) 
  {
    pos.      shfl(p.pos,   i);
    idFlt = myshfl(idFlt, p.idFlt, i);
  }

#if 0
  __device__ ParticleLight(const float4 v) :
    pos(v.x, v.y, v.z), idFlt(v.w) {}
  __device__ operator float4() const {return make_float4(pos.x, pos.y, pos.z, idFlt);}
#endif
#if 0
  __host__   int h_id() const {return __float_as_int(idFlt); }
#endif
};

template<typename T>
struct Box
{
  Position<T> centre;
  T hsize;
  __device__ Box() {}
  __device__ Box(const Position<T> &c, T hs) : centre(c), hsize(hs) {}
};

  template<typename T>
static __device__ __forceinline__ int Octant(const Position<T> &lhs, const Position<T> &rhs)
{
  return 
    ((lhs.x <= rhs.x) << 0) +
    ((lhs.y <= rhs.y) << 1) +
    ((lhs.z <= rhs.z) << 2);
};

  template<typename T>
static __device__ __forceinline__ Box<T> ChildBox(const Box<T> &box, const int oct)
{
  const T s = T(0.5) * box.hsize;
  return Box<T>(Position<T>(
        box.centre.x + s * ((oct&1) ? T(1.0) : T(-1.0)),
        box.centre.y + s * ((oct&2) ? T(1.0) : T(-1.0)),
        box.centre.z + s * ((oct&4) ? T(1.0) : T(-1.0))
        ), 
      s);
}

static __forceinline__ __device__ void computeGridAndBlockSize(dim3 &grid, dim3 &block, const int np)
{
  const int NTHREADS = 8 * WARP_SIZE;
  block = dim3(NTHREADS);
  assert(np > 0);
  grid = dim3(min(max(np/(NTHREADS*4),1), 512));
}

static __device__ __forceinline__ int lanemask_lt()
{
  int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
  return mask;
}
static __device__ __forceinline__ uint shfl_scan_add_step(const uint partial, const uint up_offset)
{
  uint result;
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.up.b32 r0|p, %1, %2, 0;"
      "@p add.u32 r0, r0, %3;"
      "mov.u32 %0, r0;}"
      : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
  return result;
}

static __device__ __forceinline__ int2 warpBinExclusiveScan(const bool p)
{
  const unsigned int b = __ballot(p);
  return make_int2(__popc(b & lanemask_lt()), __popc(b));
}

static __device__ __forceinline__ int warpBinReduce(const bool p)
{
  const unsigned int b = __ballot(p);
  return __popc(b);
}

struct CellData
{
  private:
    enum {NLEAF_SHIFT = 29};
    enum {NLEAF_MASK  = (0x1FU << NLEAF_SHIFT)};
    uint4 packed_data;
  public:
    __device__ CellData(
        const unsigned int parentCell,
        const unsigned int nBeg,
        const unsigned int nEnd,
        const unsigned int first = 0xFFFFFFFF,
        const unsigned int n = 0xFFFFFFFF)
    {
      int packed_firstleaf_n = 0xFFFFFFFF;
      if (n != 0xFFFFFFFF)
        packed_firstleaf_n = first | ((unsigned int)n << NLEAF_SHIFT);
      packed_data = make_uint4(parentCell, packed_firstleaf_n, nBeg, nEnd);
    }

    __device__ int n()      const {return packed_data.y >> NLEAF_SHIFT;}
    __device__ int first()  const {return packed_data.y  & NLEAF_MASK;}
    __device__ int parent() const {return packed_data.x;}
    __device__ int pbeg()   const {return packed_data.z;}
    __device__ int pend()   const {return packed_data.w;}

    __device__ bool isLeaf() const {return packed_data.y == 0xFFFFFFFF;}
    __device__ bool isNode() const {return !isLeaf();}
};

template<int NLEAF, typename T>
static __global__ void buildOctantSingle(
    Box<T> box,
    const int cellParentIndex,
    const int cellIndexBase,
    CellData *cellDataList,
    const int octantMask,
    __out int *octCounterBase,
    ParticleLight<T> *ptcl,
    ParticleLight<T> *buff,
    const int level = 0)
{
  typedef typename vec<4,T>::type T4;
  const int laneIdx = threadIdx.x & (WARP_SIZE-1);
  const int warpIdx = threadIdx.x >> WARP_SIZE2;

  const int octant2process = (octantMask >> (3*blockIdx.y)) & 0x7;

  int *octCounter = octCounterBase + blockIdx.y*(8+8+8+64+8);

  const int data  = octCounter[laneIdx];
  const int nCell = __shfl(data, 8+warpIdx, WARP_SIZE);
  const int nBeg  = __shfl(data, 1, WARP_SIZE);
  const int nEnd  = __shfl(data, 2, WARP_SIZE);

  int cellCounter = __shfl(data, 8+8+warpIdx, WARP_SIZE);

  /* each of the 8 warps are responsible for each of the octant */
  if (level > 0)
    box = ChildBox(box, octant2process);
  const Box<T> childBox = ChildBox(box, warpIdx);

  /* counter in shmem for each of the octant */
  int nChildren[8] = {0};

  assert(blockIdx.x == 0);

  T4* ptcl4 = (T4*)ptcl;
  T4* buff4=  (T4*)buff;

  __shared__ T4 dataX[8*WARP_SIZE];

  /* process particle array */
  dataX[threadIdx.x] = ptcl4[min(nBeg + threadIdx.x, nEnd-1)];
  __syncthreads(); 

#pragma unroll
  for (int k = 0; k < 8; k++)  /* process particles in shared memory */
  {
    if (nBeg + (k<<WARP_SIZE2) >= nEnd) break;
    const int locid = (k<<WARP_SIZE2) + laneIdx;
    const int  addr = nBeg + locid;
    const bool mask = addr < nEnd;

    const T4 p4 = dataX[locid];

#if 0          /* sanity check, check on the fly that tree structure is corrent */
    { 
      if (box.centre.x - box.hsize > p4.x ||
          box.centre.y - box.hsize > p4.y ||
          box.centre.z - box.hsize > p4.z ||
          box.centre.x + box.hsize < p4.x ||
          box.centre.y + box.hsize < p4.y ||
          box.centre.z + box.hsize < p4.z)
      {
        printf("CELL, level= %d  pos= %g %g %g   c= %g %g %g  hsize= %g\n", level,
            p4.x, p4.y,p4.z,
            box.centre.x, box.centre.y, box.centre.z, box.hsize);
        assert(0);
      }
    }
#endif

    /* use prefix sums to compute offset to where scatter particles */
    const Position<T> pos(p4.x,p4.y,p4.z);
    const int     use = mask && (Octant(box.centre, pos) == warpIdx);
    const int2 offset = warpBinExclusiveScan(use);  /* x is write offset, y is element count */

    if (offset.y > 0)
    {
      const int addrB = cellCounter;
      cellCounter += offset.y;

      int subOctant = -1;
      if (use)
      {
        buff4[addrB+offset.x] = p4;         /* float4 vector stores   */
        subOctant = Octant(childBox.centre, pos);
      }

#pragma unroll
      for (int k = 0; k < 8; k++)
        nChildren[k] += warpBinReduce(subOctant == k);
    }
  }

  __syncthreads();
  /* done processing particles, store counts atomically in gmem */
  int (*nPtclChild)[8] = (int (*)[8])dataX;

  if (laneIdx == 0)
  {
#pragma unroll
    for (int k = 0; k < 8; k++)
      nPtclChild[warpIdx][k] = nChildren[k];
  }

  /* number of particles in each cell's subcells */
  const int nSubCell = laneIdx < 8 ? nPtclChild[warpIdx][laneIdx] : 0;

  /* last block finished, analysis of the data and schedule new kernel for children */

  __syncthreads();  /* must be present, otherwise race conditions occurs between parent & children */

  int *shmem = &nPtclChild[0][0];
  if (warpIdx == 0)
    shmem[laneIdx] = 0;

  __syncthreads();

  if (threadIdx.x == 0)
    atomicCAS(&nlevels, level, level+1);

  const int nEnd1 = cellCounter;
  const int nBeg1 = nEnd1 - nCell;

  if (laneIdx == 0)
    shmem[warpIdx] = nCell;
  __syncthreads();

#if 1
  const int npCell = laneIdx < 8 ? shmem[laneIdx] : 0;

  /* compute number of children that needs to be further split, and cmopute their offsets */
  const int2 nSubNodes = warpBinExclusiveScan(npCell > NLEAF);
  const int2 nLeaves   = warpBinExclusiveScan(npCell > 0 && npCell <= NLEAF);
  if (warpIdx == 0 && laneIdx < 8)
  {
    shmem[8 +laneIdx] = nSubNodes.x;
    shmem[16+laneIdx] = nLeaves.x;
  }

  int nCellmax = npCell;
#pragma unroll
  for (int i = 2; i >= 0; i--)
    nCellmax = max(nCellmax, __shfl_xor(nCellmax, 1<<i, WARP_SIZE));

  /* if there is at least one cell to split, increment nuumber of the nodes */
  if (threadIdx.x == 0 && nSubNodes.y > 0)
  {
    shmem[16+8] = atomicAdd(&nnodes,nSubNodes.y);
#if 0   /* temp solution, a better one is to use RingBuffer */
    assert(shmem[16+8] < d_node_max);
#endif
  }

  /* writing linking info, parent, child and particle's list */
  const int nChildrenCell = warpBinReduce(laneIdx < 8 ? shmem[laneIdx] > 0 : false);
  if (threadIdx.x == 0 && nChildrenCell > 0)
  {
    const int cellFirstChildIndex = atomicAdd(&ncells, nChildrenCell);
    /*** keep in mind, the 0-level will be overwritten ***/
    const CellData cellData(cellParentIndex, nBeg, nEnd, cellFirstChildIndex, nChildrenCell);
    cellDataList[cellIndexBase + blockIdx.y] = cellData;
    shmem[16+9] = cellFirstChildIndex;
  }

  __syncthreads();
  const int cellFirstChildIndex = shmem[16+9];
  /* compute atomic data offset for cell that need to be split */
  const int next_node = shmem[16+8];
  int *octCounterNbase = &memPool[next_node*(8+8+8+64+8)];

  const int nodeOffset = shmem[8 +warpIdx];
  const int leafOffset = shmem[16+warpIdx];

  /* if cell needs to be split, populate it shared atomic data */
  if (nCell > NLEAF)
  {
    int *octCounterN = octCounterNbase + nodeOffset*(8+8+8+64+8);

    /* number of particles in each cell's subcells */
    //    const int nSubCell = laneIdx < 8 ? octCounter[8+16+warpIdx*8 + laneIdx] : 0;

    /* compute offsets */
    int cellOffset = nSubCell;
#pragma unroll
    for(int i = 0; i < 3; i++)  /* log2(8) steps */
      cellOffset = shfl_scan_add_step(cellOffset, 1 << i);
    cellOffset -= nSubCell;

    /* store offset in memory */

    cellOffset = __shfl_up(cellOffset, 8, WARP_SIZE);
    if (laneIdx < 8) cellOffset = nSubCell;
    else            cellOffset += nBeg1;
    cellOffset = __shfl_up(cellOffset, 8, WARP_SIZE);

    if (laneIdx <  8) cellOffset = 0;
    if (laneIdx == 1) cellOffset = nBeg1;
    if (laneIdx == 2) cellOffset = nEnd1;

    if (laneIdx < 24)
      octCounterN[laneIdx] = cellOffset;
  }

  /***************************/
  /*  launch  child  kernel  */
  /***************************/

  /* warps coorperate so that only 1 kernel needs to be launched by a thread block
   * with larger degree of paralellism */
  if (nSubNodes.y > 0 && warpIdx == 0)
  {
    /* build octant mask */
    int octant_mask = npCell > NLEAF ?  (laneIdx << (3*nSubNodes.x)) : 0;
#pragma unroll
    for (int i = 4; i >= 0; i--)
      octant_mask |= __shfl_xor(octant_mask, 1<<i, WARP_SIZE);

    if (threadIdx.x == 0)
    {
      dim3 grid, block;
      computeGridAndBlockSize(grid, block, nCellmax);
      hipStream_t stream;
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

      grid.y = nSubNodes.y;  /* each y-coordinate of the grid will be busy for each parent cell */
      grid.x = 1;
      buildOctantSingle<NLEAF,T><<<grid,block,0,stream>>>
        (box, cellIndexBase+blockIdx.y, cellFirstChildIndex, cellDataList,
         octant_mask, octCounterNbase, buff, ptcl, level+1);
    }
  }

  /******************/
  /* process leaves */
  /******************/

  if (nCell <= NLEAF && nCell > 0)
  {
    if (laneIdx == 0)
    {
      assert(nEnd1 - nBeg1 <= NLEAF);
      atomicAdd(&nleaves,1);
      atomicAdd(&nbodies_leaf, nEnd1-nBeg1);
      const CellData leafData(cellIndexBase+blockIdx.y, nBeg, nEnd1);
      cellDataList[cellFirstChildIndex + nSubNodes.y + leafOffset] = leafData;
    }
    if (!(level&1))
      for (int i = nBeg1+laneIdx; i < nEnd1; i += WARP_SIZE)
        if (i < nEnd1)
          ptcl4[i] = buff4[i];
  }
#endif

#if 0

  const bool isNode = shmem[laneIdx] > NLEAF;
  const int   nNode = isNode ? shmem[laneIdx] : 0;

  const int2 nSubNodes = warpBinExclusiveScan(isNode);
  if (warpIdx == 0)
    shmem[8+laneIdx] = nSubNodes.x;
  __syncthreads();

  int nCellmax = isNode ? nNode : 0;
#pragma unroll
  for (int i = 4; i >= 0; i--)
    nCellmax = max(nCellmax, __shfl_xor(nCellmax, 1<<i, WARP_SIZE));

  if (threadIdx.x == 0 && nSubNodes.y > 0)
  {
    shmem[8+8] = atomicAdd(&nnodes,nSubNodes.y);
#if 1   /* temp solution, a better one is to use RingBuffer */
    assert(shmem[8+8] < d_node_max);
#endif
  }



  /* writing linking info, parent, child and particle's list */
  const int nChildrenCell = warpBinReduce(laneIdx < 8 ? shmem[laneIdx] > 0 : false);
  if (threadIdx.x == 0 && nChildrenCell > 0)
  {
    const int cellFirstChildIndex = atomicAdd(&ncells, nChildrenCell);
    const CellData cellData(cellParentIndex, nBeg, nEnd, cellFirstChildIndex, nChildrenCell);
    cellDataList[cellIndexBase + blockIdx.y] = cellData;
    shmem[8+9] = cellFirstChildIndex;
  }

  __syncthreads();
  const int cellFirstChildIndex = shmem[8+9];
  const int next_node   = shmem[8+8];
  int *octCounterNbase  = &memPool[next_node*(8+8+8+64+8)];

  if (nCell > NLEAF)
  {
    int  *octCounterN = octCounterNbase + shmem[8+warpIdx]*(8+8+8+64+8);

    /* compute offsets */
    int cellOffset = nSubCell;
#pragma unroll
    for(int i = 0; i < 3; i++)  /* log2(8) steps */
      cellOffset = shfl_scan_add_step(cellOffset, 1 << i);
    cellOffset -= nSubCell;

    /* store offset in memory */

    cellOffset = __shfl_up(cellOffset, 8, WARP_SIZE);
    if (laneIdx < 8) cellOffset = nSubCell;
    else            cellOffset += nBeg1;
    cellOffset = __shfl_up(cellOffset, 8, WARP_SIZE);

    if (laneIdx <  8) cellOffset = 0;
    if (laneIdx == 1) cellOffset = nBeg1;
    if (laneIdx == 2) cellOffset = nEnd1;

    if (laneIdx < 24)
      octCounterN[laneIdx] = cellOffset;
  }

  /***************************/
  /*  launch  child  kernel  */
  /***************************/

  if (nSubNodes.y > 0 && warpIdx == 0)
  {
    int octant_mask = nNode > NLEAF ?  (laneIdx << (3*nSubNodes.x)) : 0;
#pragma unroll
    for (int i = 4; i >= 0; i--)
      octant_mask |= __shfl_xor(octant_mask, 1<<i, WARP_SIZE);

    if (threadIdx.x == 0)
    {
      dim3 grid, block;
      computeGridAndBlockSize(grid, block, nCellmax);
      hipStream_t stream;
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

      grid.x = 1;
      grid.y = nSubNodes.y;
      buildOctantSingle<NLEAF,T><<<grid,block,0,stream>>>
        (box, cellIndexBase+blockIdx.y, cellFirstChildIndex, cellDataList,
         octant_mask, octCounterNbase, buff, ptcl, level+1);
    }
  }

  /******************/
  /* process leaves */
  /******************/
  __syncthreads();

  const int2 nLeaves = warpBinExclusiveScan(laneIdx >= 8 ? 0 : shmem[laneIdx] > 0 && shmem[laneIdx] <= NLEAF);
  if (warpIdx == 0)
    shmem[8+laneIdx] = nLeaves.x;
  __syncthreads();

  if (shmem[warpIdx] <= NLEAF && shmem[warpIdx] > 0)
  {
    if (laneIdx == 0)
    {
      assert(nEnd1 - nBeg1 <= NLEAF);
      atomicAdd(&nleaves,1);
      atomicAdd(&nbodies_leaf, nEnd1-nBeg1);
      const CellData leafData(cellIndexBase+blockIdx.y, nBeg, nEnd1);
      cellDataList[cellFirstChildIndex + nSubNodes.y + shmem[8+warpIdx]] = leafData;
    }
    if (!(level&1))
      for (int i = nBeg1+laneIdx; i < nEnd1; i += WARP_SIZE)
        if (i < nEnd1)
          ptcl4[i] = buff4[i];
  }
#endif

}

/****** this is the main functions that build the tree recursively *******/

template<int NLEAF, typename T>
static __global__ void buildOctant(
    Box<T> box,
    const int cellParentIndex,
    const int cellIndexBase,
    CellData *cellDataList,
    const int octantMask,
    __out int *octCounterBase,
    ParticleLight<T> *ptcl,
    ParticleLight<T> *buff,
    const int level = 0)
{
  typedef typename vec<4,T>::type T4;
  /* compute laneIdx & warpIdx for each of the threads:
   *   the thread block contains only 8 warps
   *   a warp is responsible for a single octant of the cell 
   */   
  const int laneIdx = threadIdx.x & (WARP_SIZE-1);
  const int warpIdx = threadIdx.x >> WARP_SIZE2;

  /* We launch a 2D grid:
   *   the y-corrdinate carries info about which parent cell to process
   *   the x-coordinate is just a standard approach for CUDA parallelism 
   */
  const int octant2process = (octantMask >> (3*blockIdx.y)) & 0x7;

  /* get the pointer to atomic data that for a given octant */
  int *octCounter = octCounterBase + blockIdx.y*(8+8+8+64+8);

  /* read data about the current cell */
  const int data  = octCounter[laneIdx];
  const int nCell = __shfl(data, 8+warpIdx, WARP_SIZE);
  const int nBeg  = __shfl(data, 1, WARP_SIZE);
  const int nEnd  = __shfl(data, 2, WARP_SIZE);
  /* if we are not at the root level, compute the geometric box
   * of the cell */
  if (level > 0)
    box = ChildBox(box, octant2process);

  /* compute children boxes of this cell */
  const Box<T> childBox = ChildBox(box, warpIdx);

  /* countes number of particles in each octant of a child octant */
  int nChildren[8] = {0};

  /* just pointer casting to allow vector load/stores, currently works only in single precision */
  T4* ptcl4 = (T4*)ptcl;
  T4* buff4=  (T4*)buff;

  /* share storage for partiles */
  __shared__ T4 dataX[8*WARP_SIZE];

  /* process particle array */
  const int nBeg_block = nBeg + blockIdx.x * blockDim.x;
#ifdef IOCOUNT
  if (threadIdx.x == 0 && blockIdx.x == 0)
    atomicAdd(&io_words, (nEnd-nBeg)*4*sizeof(T4)/sizeof(float4));
  int nio_per_warp = 0;
#endif
  for (int i = nBeg_block; i < nEnd; i += gridDim.x * blockDim.x)
  {
    dataX[threadIdx.x] = ptcl4[min(i + threadIdx.x, nEnd-1)];
    __syncthreads(); 
#pragma unroll
    for (int k = 0; k < 8; k++)  /* process particles in shared memory */
    {
      if (i + (k<<WARP_SIZE2) >= nEnd) break;
      const int locid = (k<<WARP_SIZE2) + laneIdx;
      const int  addr = i + locid;
      const bool mask = addr < nEnd;

      const T4 p4 = dataX[locid]; //ptcl4[mask ? i+locid : nEnd-1];  /* float4 vector loads */

#if 0          /* sanity check, check on the fly that tree structure is corrent */
      { 
        if (box.centre.x - box.hsize > p4.x ||
            box.centre.y - box.hsize > p4.y ||
            box.centre.z - box.hsize > p4.z ||
            box.centre.x + box.hsize < p4.x ||
            box.centre.y + box.hsize < p4.y ||
            box.centre.z + box.hsize < p4.z)
        {
          printf("CELL, level= %d  pos= %g %g %g   c= %g %g %g  hsize= %g\n", level,
              p4.x, p4.y,p4.z,
              box.centre.x, box.centre.y, box.centre.z, box.hsize);
          assert(0);
        }
      }
#endif

      /* use prefix sums to compute offset to where scatter particles */
      const Position<T> pos(p4.x,p4.y,p4.z);
      const int     use = mask && (Octant(box.centre, pos) == warpIdx);
      const int2 offset = warpBinExclusiveScan(use);  /* x is write offset, y is element count */

      /* if this warp/octant gets particles, then write them into memory */
      if (offset.y > 0)
      {
        const int addr0 = laneIdx == 0 ? atomicAdd(&octCounter[8+8+warpIdx], offset.y) : -1;
        const int addrB = __shfl(addr0, 0, WARP_SIZE);
#ifdef IOCOUNT
        nio_per_warp += offset.y*4*sizeof(T)/sizeof(float);
#endif

        if (use)
        {
          buff4[addrB+offset.x] = p4;         /* float4 vector stores   */
          const int subOctant = Octant(childBox.centre, pos);

          switch(subOctant)  /* this way helps to unroll the nChildren into registers */
          {
            case 0: nChildren[0]++; break;
            case 1: nChildren[1]++; break;
            case 2: nChildren[2]++; break;
            case 3: nChildren[3]++; break;
            case 4: nChildren[4]++; break;
            case 5: nChildren[5]++; break;
            case 6: nChildren[6]++; break;
            case 7: nChildren[7]++; break;
          };
        }
      }
    }
    __syncthreads(); 
  }

  /* done processing particles, store number of particle in each octant of a child cell */

  int (*nPtclChild)[8] = (int (*)[8])dataX;

#ifdef IOCOUNT
  nPtclChild[0][warpIdx] = nio_per_warp;
  __syncthreads();
  nio_per_warp = laneIdx < 8 ? nPtclChild[0][laneIdx] : 0;
#pragma unroll
  for (int i = 2; i >= 0; i--)
    nio_per_warp += __shfl_xor(nio_per_warp, 1<<i, WARP_SIZE);
  if (threadIdx.x == 0)
    atomicAdd(&io_words, nio_per_warp);
#endif

#pragma unroll
  for (int i = 4; i >= 0; i--)
  {
#pragma unroll
    for (int k = 0; k < 8; k++)
      nChildren[k] += __shfl_xor(nChildren[k], 1<<i, WARP_SIZE);
  }

  if (laneIdx == 0)
  {
#pragma unroll
    for (int k = 0; k < 8; k++)
      nPtclChild[warpIdx][k] = nChildren[k];
  }
  if (laneIdx < 8)
    if (nPtclChild[warpIdx][laneIdx] > 0)
      atomicAdd(&octCounter[8+16+warpIdx*8 + laneIdx], nPtclChild[warpIdx][laneIdx]);

  __syncthreads();  /* must be present, otherwise race conditions occurs between parent & children */

  /* detect last thread block for unique y-coordinate of the grid:
   * mind, this cannot be done on the host, because we don't detect last 
   * block on the grid, but instead the last x-block for each of the y-coordainte of the grid
   * this should increase the degree of parallelism
   */

  int *shmem = &nPtclChild[0][0];
  if (warpIdx == 0)
    shmem[laneIdx] = 0;

  int &lastBlock = shmem[0];
  if (threadIdx.x == 0)
  {
    const int ticket = atomicAdd(octCounter, 1);
    lastBlock = (ticket == gridDim.x-1);
  }
  __syncthreads();

  if (!lastBlock) return;

  __syncthreads();

  /* okay, we are in the last thread block, do the analysis and decide what to do next */

  if (warpIdx == 0)
    shmem[laneIdx] = 0;

  if (threadIdx.x == 0)
    atomicCAS(&nlevels, level, level+1);

  __syncthreads();

  /* compute beginning and then end addresses of the sorted particles  in the child cell */
  const int nEnd1 = octCounter[8+8+warpIdx];
  const int nBeg1 = nEnd1 - nCell;

  if (laneIdx == 0)
    shmem[warpIdx] = nCell;
  __syncthreads();

  const int npCell = laneIdx < 8 ? shmem[laneIdx] : 0;

  /* compute number of children that needs to be further split, and cmopute their offsets */
  const int2 nSubNodes = warpBinExclusiveScan(npCell > NLEAF);
  const int2 nLeaves   = warpBinExclusiveScan(npCell > 0 && npCell <= NLEAF);
  if (warpIdx == 0 && laneIdx < 8)
  {
    shmem[8 +laneIdx] = nSubNodes.x;
    shmem[16+laneIdx] = nLeaves.x;
  }

  int nCellmax = npCell;
#pragma unroll
  for (int i = 2; i >= 0; i--)
    nCellmax = max(nCellmax, __shfl_xor(nCellmax, 1<<i, WARP_SIZE));

  /* if there is at least one cell to split, increment nuumber of the nodes */
  if (threadIdx.x == 0 && nSubNodes.y > 0)
  {
    shmem[16+8] = atomicAdd(&nnodes,nSubNodes.y);
#if 0   /* temp solution, a better one is to use RingBuffer */
    assert(shmem[16+8] < d_node_max);
#endif
  }

  /* writing linking info, parent, child and particle's list */
  const int nChildrenCell = warpBinReduce(laneIdx < 8 ? shmem[laneIdx] > 0 : false);
  if (threadIdx.x == 0 && nChildrenCell > 0)
  {
    const int cellFirstChildIndex = atomicAdd(&ncells, nChildrenCell);
    /*** keep in mind, the 0-level will be overwritten ***/
    const CellData cellData(cellParentIndex, nBeg, nEnd, cellFirstChildIndex, nChildrenCell);
    cellDataList[cellIndexBase + blockIdx.y] = cellData;
    shmem[16+9] = cellFirstChildIndex;
  }

  __syncthreads();
  const int cellFirstChildIndex = shmem[16+9];
  /* compute atomic data offset for cell that need to be split */
  const int next_node = shmem[16+8];
  int *octCounterNbase = &memPool[next_node*(8+8+8+64+8)];

  const int nodeOffset = shmem[8 +warpIdx];
  const int leafOffset = shmem[16+warpIdx];

  /* if cell needs to be split, populate it shared atomic data */
  if (nCell > NLEAF)
  {
    int *octCounterN = octCounterNbase + nodeOffset*(8+8+8+64+8);

    /* number of particles in each cell's subcells */
    const int nSubCell = laneIdx < 8 ? octCounter[8+16+warpIdx*8 + laneIdx] : 0;

    /* compute offsets */
    int cellOffset = nSubCell;
#pragma unroll
    for(int i = 0; i < 3; i++)  /* log2(8) steps */
      cellOffset = shfl_scan_add_step(cellOffset, 1 << i);
    cellOffset -= nSubCell;

    /* store offset in memory */

    cellOffset = __shfl_up(cellOffset, 8, WARP_SIZE);
    if (laneIdx < 8) cellOffset = nSubCell;
    else            cellOffset += nBeg1;
    cellOffset = __shfl_up(cellOffset, 8, WARP_SIZE);

    if (laneIdx <  8) cellOffset = 0;
    if (laneIdx == 1) cellOffset = nBeg1;
    if (laneIdx == 2) cellOffset = nEnd1;

    if (laneIdx < 24)
      octCounterN[laneIdx] = cellOffset;
  }

  /***************************/
  /*  launch  child  kernel  */
  /***************************/

  /* warps coorperate so that only 1 kernel needs to be launched by a thread block
   * with larger degree of paralellism */
  if (nSubNodes.y > 0 && warpIdx == 0)
  {
    /* build octant mask */
    int octant_mask = npCell > NLEAF ?  (laneIdx << (3*nSubNodes.x)) : 0;
#pragma unroll
    for (int i = 4; i >= 0; i--)
      octant_mask |= __shfl_xor(octant_mask, 1<<i, WARP_SIZE);

    if (threadIdx.x == 0)
    {
      dim3 grid, block;
      computeGridAndBlockSize(grid, block, nCellmax);
      hipStream_t stream;
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

      grid.y = nSubNodes.y;  /* each y-coordinate of the grid will be busy for each parent cell */
      if (nCellmax <= block.x)
      {
        grid.x = 1;
        buildOctantSingle<NLEAF,T><<<grid,block,0,stream>>>
          (box, cellIndexBase+blockIdx.y, cellFirstChildIndex, cellDataList,
           octant_mask, octCounterNbase, buff, ptcl, level+1);
      }
      else
      {
        buildOctant<NLEAF,T><<<grid,block,0,stream>>>
          (box, cellIndexBase+blockIdx.y, cellFirstChildIndex, cellDataList,
           octant_mask, octCounterNbase, buff, ptcl, level+1);
      }
    }
  }

  /******************/
  /* process leaves */
  /******************/

  if (nCell <= NLEAF && nCell > 0)
  {
    if (laneIdx == 0)
    {
      assert(nEnd1 - nBeg1 <= NLEAF);
      atomicAdd(&nleaves,1);
      atomicAdd(&nbodies_leaf, nEnd1-nBeg1);
      const CellData leafData(cellIndexBase+blockIdx.y, nBeg, nEnd1);
      cellDataList[cellFirstChildIndex + nSubNodes.y + leafOffset] = leafData;
    }
    if (!(level&1))
      for (int i = nBeg1+laneIdx; i < nEnd1; i += WARP_SIZE)
        if (i < nEnd1)
          ptcl4[i] = buff4[i];
  }
}

/******* compute multipole moments ****/

  template<int NTHREADS>
static __device__ double reduceBlock(double sum)
{
  extern volatile __shared__ double sh[];
  const int tid = threadIdx.x;

  sh[tid] = sum;
  __syncthreads();

  if (NTHREADS >= 512)
  {
    if (tid < 256) sum = sh[tid] = sum + sh[tid + 256];
    __syncthreads();
  }
  if (NTHREADS >= 256)
  {
    if (tid < 128) sum = sh[tid] = sum + sh[tid + 128];
    __syncthreads();
  }
  if (NTHREADS >= 128)
  {
    if (tid < 64) sum = sh[tid] = sum + sh[tid + 64];
    __syncthreads();
  }
  if (tid < 32)
  {
    if (NTHREADS >= 64)  sum = sh[tid] = sum + sh[tid + 32];
    if (NTHREADS >= 32)  sum = sh[tid] = sum + sh[tid + 16];
    if (NTHREADS >= 16)  sum = sh[tid] = sum + sh[tid +  8];
    if (NTHREADS >=  8)  sum = sh[tid] = sum + sh[tid +  4];
    if (NTHREADS >=  4)  sum = sh[tid] = sum + sh[tid +  2];
    if (NTHREADS >=  2)  sum = sh[tid] = sum + sh[tid +  1];
  }
  __syncthreads();
  return sh[0];
};

template<int NTHREADS, typename T>
static __global__ 
void computeNodeProperties(
    const int n,
    const CellData     *cellDataList,
    const Particle4<T> *ptclPosList,
    typename vec<4,T>::type *cellCOM,
    typename vec<4,T>::type *cellQMxx_yy_zz_m,
    typename vec<4,T>::type *cellQMxy_xz_yz)
{
  typedef typename vec<4,T>::type T4;
  const int cellIdx = blockIdx.x;
  const CellData cellData = cellDataList[cellIdx];

  double4 monopoleM = {0.0, 0.0, 0.0, 0.0};
  double3 Qxx_yy_zz = {0.0, 0.0, 0.0};
  double3 Qxy_xz_yz = {0.0, 0.0, 0.0};
  for (int i = cellData.pbeg(); i < cellData.pend(); i += blockDim.x)
  {
    const bool mask = (i + threadIdx.x) < cellData.pend();
    Particle4<T> ip;
    if (mask) ip = ptclPosList[i + threadIdx.x];

    double mass = mask ? ip.mass() : 0.0; 
    double3 pos = make_double3(ip.x(), ip.y(), ip.z());

    monopoleM.x += mass * pos.x;
    monopoleM.y += mass * pos.y;
    monopoleM.z += mass * pos.z;
    monopoleM.w += mass;

    Qxx_yy_zz.x += mass * pos.x*pos.x;
    Qxx_yy_zz.y += mass * pos.y*pos.y;
    Qxx_yy_zz.z += mass * pos.z*pos.z;

    Qxy_xz_yz.x += mass * pos.x*pos.y;
    Qxy_xz_yz.y += mass * pos.x*pos.z;
    Qxy_xz_yz.z += mass * pos.y*pos.z;
  }

#if 0
  monopoleM.x = reduceBlock<NTHREADS>(monopoleM.x); __syncthreads();
  monopoleM.y = reduceBlock<NTHREADS>(monopoleM.y); __syncthreads();
  monopoleM.z = reduceBlock<NTHREADS>(monopoleM.z); __syncthreads();
  monopoleM.w = reduceBlock<NTHREADS>(monopoleM.w); __syncthreads();

  Qxx_yy_zz.x = reduceBlock<NTHREADS>(Qxx_yy_zz.x); __syncthreads();
  Qxx_yy_zz.y = reduceBlock<NTHREADS>(Qxx_yy_zz.y); __syncthreads();
  Qxx_yy_zz.z = reduceBlock<NTHREADS>(Qxx_yy_zz.z); __syncthreads();

  Qxy_xz_yz.x = reduceBlock<NTHREADS>(Qxy_xz_yz.x); __syncthreads();
  Qxy_xz_yz.y = reduceBlock<NTHREADS>(Qxy_xz_yz.y); __syncthreads();
  Qxy_xz_yz.z = reduceBlock<NTHREADS>(Qxy_xz_yz.z); __syncthreads();
#endif

  //  assert(monopoleM.w > 0.0);
  const double invMass = monopoleM.w;

  T4 icellCOM;
  icellCOM.x = T(monopoleM.x * invMass);
  icellCOM.y = T(monopoleM.y * invMass);
  icellCOM.z = T(monopoleM.z * invMass);
  icellCOM.w = -1.0;
  if (threadIdx.x == 0)
    cellCOM[cellIdx] = icellCOM;

  T4 icellQxx_yy_zz_m;
  icellQxx_yy_zz_m.x = T(Qxx_yy_zz.x);
  icellQxx_yy_zz_m.y = T(Qxx_yy_zz.y);
  icellQxx_yy_zz_m.z = T(Qxx_yy_zz.z);
  icellQxx_yy_zz_m.w = T(monopoleM.w);
  if (threadIdx.x == 0)
    cellQMxx_yy_zz_m[cellIdx] = icellQxx_yy_zz_m;

  T4 icellQxy_xz_yz;
  icellQxy_xz_yz.x = T(Qxy_xz_yz.x);
  icellQxy_xz_yz.y = T(Qxy_xz_yz.y);
  icellQxy_xz_yz.z = T(Qxy_xz_yz.z);
  icellQxy_xz_yz.w = T(0.0);
  if (threadIdx.x == 0)
    cellQMxy_xz_yz[cellIdx] = icellQxy_xz_yz;
}


/****** not very tuned kernels to do preparatory stuff ********/


template<typename T, const int NTHREADS>
static __device__ void reduceBlock(
    volatile Position<T> *shmin,
    volatile Position<T> *shmax,
    Position<T> bmin,
    Position<T> bmax)
{
  const int tid = threadIdx.x;

#define STORE {\
  shmin[tid].x = bmin.x; shmin[tid].y = bmin.y; shmin[tid].z = bmin.z; \
  shmax[tid].x = bmax.x; shmax[tid].y = bmax.y; shmax[tid].z = bmax.z; }

  STORE;
  __syncthreads();

  // do reduction in shared mem
  if (NTHREADS >= 512)
  {
    if (tid < 256)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+256]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+256]));
      STORE;
    }
    __syncthreads();
  }

  if (NTHREADS >= 256)
  {
    if (tid < 128)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+128]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+128]));
      STORE;
    }
    __syncthreads();
  }

  if (NTHREADS >= 128)
  {
    if (tid <  64)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+64]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+64]));
      STORE;
    }
    __syncthreads();
  }


  if (tid < 32)
  {
    if (NTHREADS >=  64)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+32]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+32]));
      STORE;
    }
    if (NTHREADS >=  32)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+16]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+16]));
      STORE;
    }
    if (NTHREADS >=  16)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+8]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+8]));
      STORE;
    }
    if (NTHREADS >=   8)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+4]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+4]));
      STORE;
    }
    if (NTHREADS >=   4)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+2]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+2]));
      STORE;
    }
    if (NTHREADS >=   2)
    {
      bmin = Position<T>::min(bmin, get_volatile<T>(shmin[tid+1]));
      bmax = Position<T>::max(bmax, get_volatile<T>(shmax[tid+1]));
      STORE;
    }
  }
#undef STORE

  __syncthreads();
}

template<typename T, const int NBLOCKS, const int NTHREADS>
static __global__ void computeBoundingBox(
    const int n,
    __out Position<T> *minmax_ptr,
    __out Box<T>      *box_ptr,
    __out int *retirementCount,
    const ParticleLight<T> *ptcl)
{
  __shared__ Position<T> shmin[NTHREADS], shmax[NTHREADS];

  const int gridSize = NTHREADS*NBLOCKS*2;
  int i = blockIdx.x*NTHREADS*2 + threadIdx.x;

  Position<T> bmin(T(+1e10)), bmax(T(-1e10));

  while (i < n)
  {
    const ParticleLight<T> p = ptcl[i];
    bmin = Position<T>::min(bmin, p.pos);
    bmax = Position<T>::max(bmax, p.pos);
    if (i + NTHREADS < n)
    {
      const ParticleLight<T> p = ptcl[i + NTHREADS];
      bmin = Position<T>::min(bmin, p.pos);
      bmax = Position<T>::max(bmax, p.pos);
    }
    i += gridSize;
  }

  reduceBlock<T, NTHREADS>(shmin, shmax, bmin, bmax);
  if (threadIdx.x == 0) 
  {
    bmin = shmin[0];
    bmax = shmax[0];
    minmax_ptr[blockIdx.x          ] = bmin;
    minmax_ptr[blockIdx.x + NBLOCKS] = bmax;
  }

  __shared__ bool lastBlock;
  __threadfence();

  if (threadIdx.x == 0)
  {
    const int ticket = atomicInc((unsigned int*)retirementCount, NBLOCKS);
    lastBlock = (ticket == NBLOCKS - 1);
  }

  __syncthreads();

  if (lastBlock)
  {
    Position<T> bmin(T(+1e10)), bmax(T(-1e10));
    int i = threadIdx.x;
    while (i < NBLOCKS)
      if (i < NBLOCKS)
      {
        bmin = Position<T>::min(bmin, minmax_ptr[i        ]);
        bmax = Position<T>::max(bmax, minmax_ptr[i+NBLOCKS]);
        i += NTHREADS;
      };

    reduceBlock<T, NTHREADS>(shmin, shmax, bmin, bmax);
    __syncthreads();

    if (threadIdx.x == 0)
    {
      bmin = shmin[0];
      bmax = shmax[0];
      const Position<T> cvec((bmax.x+bmin.x)*T(0.5), (bmax.y+bmin.y)*T(0.5), (bmax.z+bmin.z)*T(0.5));
      const Position<T> hvec((bmax.x-bmin.x)*T(0.5), (bmax.y-bmin.y)*T(0.5), (bmax.z-bmin.z)*T(0.5));
      const T h = fmax(hvec.z, fmax(hvec.y, hvec.x));
      T hsize = T(1.0);
      while (hsize > h) hsize *= T(0.5);
      while (hsize < h) hsize *= T(2.0);
#if 0
      hsize *= T(128.0);
#endif

      const int NMAXLEVEL = 20;

      const T hquant = hsize / T(1<<NMAXLEVEL);
      const long long nx = (long long)(cvec.x/hquant);
      const long long ny = (long long)(cvec.y/hquant);
      const long long nz = (long long)(cvec.z/hquant);

      const Position<T> centre(hquant * T(nx), hquant * T(ny), hquant * T(nz));

      *box_ptr = Box<T>(centre, hsize);
    };
  }
}


template<typename T>
static __global__ void countAtRootNode(
    const int n,
    __out int *octCounter,
    const Box<T> box,
    const ParticleLight<T> *ptcl)
{
  const int beg = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = beg; i < n; i += gridDim.x * blockDim.x)
    if (i < n)
    {
      const ParticleLight<T> p = ptcl[i];
      const int octant = Octant(box.centre, p.pos);
      atomicAdd(&octCounter[8+octant],1);
    };

#if 0
  __shared__ bool lastBlock;
  __threadfence();
  if (threadIdx.x == 0)
  {
    const int ticket = atomicInc((unsigned int*)octCounter, gridDim.x);
    lastBlock = (ticket == gridDim.x-1);
  };

  __syncthreads();
#endif
}

template<int NLEAF, typename T>
static __global__ void buildOctree(
    const int n,
    CellData *cellDataList,
    int* memory_pool,
    __out ParticleLight<T> *ptcl,
    __out ParticleLight<T> *buff,
    int *ncells_ret = NULL)
{
  typedef typename vec<4,T>::type T4;
  memPool = memory_pool;
  printf("d_node_max= %d\n", d_node_max);
  printf("d_cell_max= %d\n", d_cell_max);
  const int NTHREADS = 256;
  const int NBLOCKS  = 256;
  Box<T> *box_ptr = new Box<T>();
  Position<T> *minmax_ptr = new Position<T>[2*NBLOCKS];
  int *retirementCount = new int;
  *retirementCount = 0;
  __threadfence();
  computeBoundingBox<T,NBLOCKS,NTHREADS><<<NBLOCKS,NTHREADS>>>(n, minmax_ptr, box_ptr, retirementCount, ptcl);
  hipDeviceSynchronize();
  delete retirementCount;

  printf("GPU: box_centre= %g %g %g   hsize= %g\n",
      box_ptr->centre.x,
      box_ptr->centre.y,
      box_ptr->centre.z,
      box_ptr->hsize);

  int *octCounter = new int[8+8];
  for (int k = 0; k < 16; k++)
    octCounter[k] = 0;
  countAtRootNode<T><<<256, 256>>>(n, octCounter, *box_ptr, ptcl);
  hipDeviceSynchronize();

  int total = 0;
  for (int k = 8; k < 16; k++)
  {
    printf("octCounter[%d]= %d\n", k-8, octCounter[k]);
    total += octCounter[k];
  }
  printf("total= %d  n= %d\n", total, n);

  int *octCounterN = new int[8+8+8+64+8];
#pragma unroll
  for (int k = 0; k < 8; k++)
  {
    octCounterN[     k] = 0;
    octCounterN[8+   k] = octCounter[8+k  ];
    octCounterN[8+8 +k] = k == 0 ? 0 : octCounterN[8+8+k-1] + octCounterN[8+k-1];
    octCounterN[8+16+k] = 0;
  }
#pragma unroll
  for (int k = 8; k < 64; k++)
    octCounterN[8+16+k] = 0;

  for (int k = 0; k < 8; k++)
    printf("k= %d n = %d offset= %d \n",
        k, octCounterN[8+k], octCounterN[8+8+k]);

  io_words = 0;
  nnodes = 0;
  nleaves = 0;
  nlevels = 0;
  ncells  = 0;
  nbodies_leaf = 0;

  octCounterN[1] = 0;
  octCounterN[2] = n;



  dim3 grid, block;
  computeGridAndBlockSize(grid, block, n);
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  buildOctant<NLEAF,T><<<grid, block,0,stream>>>
    (box_ptr[0], 0, 0, cellDataList, 0, octCounterN, ptcl, buff);
  hipDeviceSynchronize();

  printf(" nptcl  = %d\n", n);
  printf(" nb_leaf= %d\n", nbodies_leaf);
  printf(" nnodes = %d\n", nnodes);
  printf(" nleaves= %d\n", nleaves);
  printf(" ncells=  %d\n",  ncells);
  if (ncells_ret != NULL)
    *ncells_ret = ncells;
  printf(" nlevels= %d\n", nlevels);

#ifdef IOCOUNT
  printf(" io= %g MB \n" ,io_words*4.0/1024.0/1024.0);
#endif

  delete [] octCounter;
  delete [] minmax_ptr;
  delete [] octCounterN;
  delete box_ptr;
}

template<typename T>
  static __global__
void moveParticles(const int n, 
    const ParticleLight<T> *ptcl_idx,
    const Particle4<T> *ptclIn, Particle4<T> *ptclOut)
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= n) return;
  const ParticleLight<T> &pIdx = ptcl_idx[idx];
  ptclOut[idx] = ptclIn[pIdx.id()];
}

int main(int argc, char * argv [])
{
  int n = 4000000;
  if (argc > 1)
  {
    assert(argc > 1);
    n = atoi(argv[1]);
  }
  assert(n > 0);

  fprintf(stderr, " n= %d \n", n);

#ifdef FP64
  typedef double real;
#else
  typedef float real;
#endif

  typedef typename vec<4, real>::type real4;

  host_mem< ParticleLight<real> > h_ptcl;
  host_mem< Particle4<real> > h_ptclPos;
  h_ptcl.alloc(n);
  h_ptclPos.alloc(n);
#ifdef PLUMMER
  const Plummer data(n, argc > 2 ? atoi(argv[2]) : 19810614);
  for (int i = 0; i < n; i++)
  {
    h_ptcl[i] = ParticleLight<real>(Position<real>(data.pos[i].x, data.pos[i].y, data.pos[i].z), i);
    h_ptclPos[i].x()    = data.pos[i].x;
    h_ptclPos[i].y()    = data.pos[i].y;
    h_ptclPos[i].z()    = data.pos[i].z;
    h_ptclPos[i].mass() = 1.0/n;
  }
#else
  for (int i = 0; i < n; i++)
  {
    h_ptcl[i] = ParticleLight<real>(Position<real>(drand48(), drand48(), drand48()), i);
    h_ptclPos[i].x()    = h_ptcl[i].x;
    h_ptclPos[i].y()    = h_ptcl[i].y;
    h_ptclPos[i].z()    = h_ptcl[i].z;
    h_ptclPos[i].mass() = 1.0/n;
  }
#endif
  Position<real> bmin(+1e10), bmax(-1e10);
  for (int i = 0; i < n; i++)
  {
    //    printf("%g %g %g \n", h_ptcl[i].pos.x, h_ptcl[i].pos.y, h_ptcl[i].pos.z);
    bmin = Position<real>::min(bmin, h_ptcl[i].pos);
    bmax = Position<real>::max(bmax, h_ptcl[i].pos);
  }
  //  exit(0);
  const Position<real> cvec((bmax.x+bmin.x)*(0.5), (bmax.y+bmin.y)*(0.5), (bmax.z+bmin.z)*(0.5));
  const Position<real> hvec((bmax.x-bmin.x)*(0.5), (bmax.y-bmin.y)*(0.5), (bmax.z-bmin.z)*(0.5));
  const real h = fmax(hvec.z, fmax(hvec.y, hvec.x));
  real hsize = (1.0);
  while (hsize > h) hsize *= (0.5);
  while (hsize < h) hsize *= (2.0);

  fprintf(stderr, "bmin= %g %g %g \n", bmin.x, bmin.y, bmin.z);
  fprintf(stderr, "bmax= %g %g %g \n", bmax.x, bmax.y, bmax.z);

  printf("box_centre= %g %g %g   hsize= %g\n",
      cvec.x,
      cvec.y,
      cvec.z,
      hsize);

  cuda_mem< ParticleLight<real> > d_ptcl1, d_ptcl2;
  cuda_mem< Particle4<real> > d_ptclPos;
  d_ptcl1.alloc(n);
  d_ptcl2.alloc(n);
  d_ptclPos.alloc(n);
  d_ptcl1.h2d(h_ptcl);
  d_ptclPos.h2d(h_ptclPos);


  int node_max = n/10;
  int cell_max = n;

  cuda_mem<int> memory_pool;
  const unsigned long long nstack = (8+8+8+64+8)*node_max;
  fprintf(stderr, " nstack= %g MB \n", sizeof(int)*nstack/1024.0/1024.0);

  memory_pool.alloc(nstack);
  CUDA_SAFE_CALL(hipMemset(memory_pool,0,nstack*sizeof(int)));

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_node_max), &node_max, sizeof(int), 0, hipMemcpyHostToDevice));


  cuda_mem<CellData> cellDataList;
  cellDataList.alloc(cell_max);
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_cell_max), &cell_max, sizeof(int), 0, hipMemcpyHostToDevice));


#ifndef NPERLEAF
  const int NLEAF = 16;
#else
  const int NLEAF = NPERLEAF;
#endif

#if 0
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&buildOctant<NLEAF), real>, hipFuncCachePreferL1));
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&buildOctantSingle<NLEAF), real>, hipFuncCachePreferL1));
#elif 1
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&buildOctant<NLEAF), real>, hipFuncCachePreferShared));
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&buildOctantSingle<NLEAF), real>, hipFuncCachePreferShared));
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&computeNodeProperties<64), real>, hipFuncCachePreferShared));
#else
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&buildOctant<NLEAF), real>, hipFuncCachePreferEqual));
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&buildOctantSingle<NLEAF), real>, hipFuncCachePreferEqual));
#endif

  {
    const double t0 = rtc();
    buildOctree<NLEAF, real><<<1,1>>>(n, cellDataList, memory_pool, d_ptcl1, d_ptcl2);
    const int ret = (hipDeviceSynchronize() != hipSuccess);
    if (ret)
    {
      fprintf(stderr, "CNP tree launch failed: %s\n", hipGetErrorString(hipGetLastError()));
      assert(0);
    }

    const double t1 = rtc();
    const double dt = t1 - t0;

    fprintf(stderr, " done in %g sec : %g Mptcl/sec\n",
        dt, n/1e6/dt);
  }

  CUDA_SAFE_CALL(hipMemset(memory_pool,0,nstack*sizeof(int)));

  {
    const double t0 = rtc();
    cuda_mem<int> d_ncells;
    host_mem<int> h_ncells;
    d_ncells.alloc(1);
    h_ncells.alloc(1);
    buildOctree<NLEAF, real><<<1,1>>>(n, cellDataList, memory_pool, d_ptcl1, d_ptcl2, d_ncells);
    const int ret = (hipDeviceSynchronize() != hipSuccess);
    if (ret)
      fprintf(stderr, "CNP tree launch failed: %s\n", hipGetErrorString(hipGetLastError()));

    const double t1 = rtc();
    const double dt = t1 - t0;

    fprintf(stderr, " tree done in %g sec : %g Mptcl/sec\n",
        dt, n/1e6/dt);

    d_ncells.d2h(h_ncells);
    const int ncells = h_ncells[0];
    fprintf(stderr, "ncells= %d\n", ncells);

    cuda_mem<real4> d_cellCOM, d_cellQMxx_yy_zz_m, d_cellQMxy_xz_yz;
    d_cellCOM.alloc(ncells);
    d_cellQMxx_yy_zz_m.alloc(ncells);
    d_cellQMxy_xz_yz.alloc(ncells);

    hipDeviceSynchronize();
    {
      const double t0 = rtc();
      const int NTHREADS=128;
      computeNodeProperties<NTHREADS,real><<<ncells,NTHREADS,sizeof(double)*NTHREADS>>>(
          n,
          cellDataList,
          d_ptclPos,
          d_cellCOM,
          d_cellQMxx_yy_zz_m,
          d_cellQMxy_xz_yz);
      const int ret = (hipDeviceSynchronize() != hipSuccess);
      if (ret)
      {
        fprintf(stderr, "computeNodePorperties launch failed: %s\n", hipGetErrorString(hipGetLastError()));
        assert(0);
      }
      const double t1 = rtc();
      const double dt = t1 - t0;

      fprintf(stderr, " properties done in %g sec : %g Mptcl/sec\n",
          dt, n/1e6/dt);
    }


  }

};

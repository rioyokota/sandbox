#include "hip/hip_runtime.h"
#include "serialfmm.h"

int main() {
  double tic, toc;
  int numBodies = 1000;
  THETA = 0.6;
  Bodies bodies;
  Cells cells;
  SerialFMM FMM;
  for( int it=0; it<25; ++it ) {
    numBodies = int(pow(10,(it+24)/8.0));
    std::cout << "N                    : " << numBodies << std::endl;
    bodies.resize(numBodies);
    FMM.dataset(bodies);

    tic = FMM.getTime();
    FMM.bottomup(bodies,cells);
    FMM.evaluate(cells);
    toc = FMM.getTime();
    if( FMM.printNow ) printf("FMM                  : %lf\n",toc-tic);

    tic = FMM.getTime();
    FMM.direct(bodies);
    toc = FMM.getTime();
    if( FMM.printNow ) printf("Direct               : %lf\n",toc-tic);
  }
}

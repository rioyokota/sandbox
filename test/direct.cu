#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <papi.h>
#include <sys/time.h>

const int THREADS = 512;
const int N = THREADS * 128;
const float OPS = 20. * N * N * 1e-9;
const float EPS2 = 1e-6;

double get_time() {
  struct timeval tv;
  hipDeviceSynchronize();
  gettimeofday(&tv,NULL);
  return double(tv.tv_sec+tv.tv_usec*1e-6);
}

extern void P2Psse(float4 *target, float4 *source, int ni, int nj, float eps2);

extern void P2Pasm(float4 *target, float4 *source, int ni, int nj, float eps2);

__global__ void P2Pdevice(float4 *target, float4 *source) {
  int i = blockIdx.x * THREADS + threadIdx.x;
  float4 t = {0,0,0,0};
  __shared__ float4 s[THREADS];
  for ( int jb=0; jb<N/THREADS; jb++ ) {
    __syncthreads();
    s[threadIdx.x] = source[jb*THREADS+threadIdx.x];
    __syncthreads();
    for( int j=0; j<THREADS; j++ ) {
      float dx = s[j].x - source[i].x;
      float dy = s[j].y - source[i].y;
      float dz = s[j].z - source[i].z;
      float R2 = dx * dx + dy * dy + dz * dz + EPS2;
      float invR = rsqrtf(R2);
      t.w += s[j].w * invR;
      float invR3 = invR * invR * invR * s[j].w;
      t.x += dx * invR3;
      t.y += dy * invR3;
      t.z += dz * invR3;
    }
  }
  target[i] = t;
}

int main() {
// ALLOCATE
  float4 *sourceHost = new float4 [N];
  float4 *targetSSE = new float4 [N];
  float4 *targetASM = new float4 [N];
  float4 *targetGPU = new float4 [N];
  for( int i=0; i<N; i++ ) {
    sourceHost[i].x = drand48();
    sourceHost[i].y = drand48();
    sourceHost[i].z = drand48();
    sourceHost[i].w = drand48() / N;
  }
  float4 *sourceDevc, *targetDevc;
  hipMalloc((void**)&sourceDevc,N*sizeof(float4));
  hipMalloc((void**)&targetDevc,N*sizeof(float4));
  std::cout << std::scientific << "N     : " << N << std::endl;

// SSE P2P
  int Events[3] = { PAPI_L2_DCM, PAPI_L2_DCA, PAPI_TLB_DM };
  int EventSet = PAPI_NULL;
  PAPI_library_init(PAPI_VER_CURRENT);
  PAPI_create_eventset(&EventSet);
  PAPI_add_events(EventSet, Events, 3);
  PAPI_start(EventSet);

  double tic = get_time();
  P2Psse(targetSSE,sourceHost,N,N,EPS2);
  double toc = get_time();

  long long values[3];
  PAPI_stop(EventSet,values);
  std::cout << "L2 Miss: " << values[0]
            << " L2 Access: " << values[1]
            << " TLB Miss: " << values[2] << std::endl;

  std::cout << std::scientific << "SSE   : " << toc-tic << " s : " << OPS / (toc-tic) << " GFlops" << std::endl;

// SSE P2P
  PAPI_start(EventSet);

  tic = get_time();
  P2Pasm(targetASM,sourceHost,N,N,EPS2);
  toc = get_time();

  PAPI_stop(EventSet,values);
  std::cout << "L2 Miss: " << values[0]
            << " L2 Access: " << values[1]
            << " TLB Miss: " << values[2] << std::endl;

  std::cout << std::scientific << "ASM   : " << toc-tic << " s : " << OPS / (toc-tic) << " GFlops" << std::endl;

// GPU P2P
  hipMemcpy(sourceDevc,sourceHost,N*sizeof(float4),hipMemcpyHostToDevice);
  tic = get_time();
  P2Pdevice<<<N/THREADS,THREADS>>>(targetDevc,sourceDevc);
  toc = get_time();
  hipMemcpy(targetGPU,targetDevc,N*sizeof(float4),hipMemcpyDeviceToHost);
  std::cout << std::scientific << "GPU   : " << toc-tic << " s : " << OPS / (toc-tic) << " GFlops" << std::endl;
  hipDeviceReset();

// COMPARE RESULTS
  float pd = 0, pn = 0, fd = 0, fn = 0;
  for( int i=0; i<N; i++ ) {
    targetSSE[i].w -= sourceHost[i].w / sqrtf(EPS2);
    targetGPU[i].w -= sourceHost[i].w / sqrtf(EPS2);
    pd += (targetSSE[i].w - targetGPU[i].w) * (targetSSE[i].w - targetGPU[i].w);
    pn += targetSSE[i].w * targetSSE[i].w;
    fd += (targetSSE[i].x - targetGPU[i].x) * (targetSSE[i].x - targetGPU[i].x)
        + (targetSSE[i].y - targetGPU[i].y) * (targetSSE[i].y - targetGPU[i].y)
        + (targetSSE[i].z - targetGPU[i].z) * (targetSSE[i].z - targetGPU[i].z);
    fn += targetSSE[i].x * targetSSE[i].x + targetSSE[i].y * targetSSE[i].y + targetSSE[i].z * targetSSE[i].z;
  }
  std::cout << std::scientific << "P ERR : " << sqrtf(pd/pn) << std::endl;
  std::cout << std::scientific << "F ERR : " << sqrtf(fd/fn) << std::endl;

// DEALLOCATE
  hipFree(sourceDevc);
  hipFree(targetDevc);
  delete[] sourceHost;
  delete[] targetSSE;
  delete[] targetASM;
  delete[] targetGPU;
}

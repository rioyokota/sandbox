#include "hip/hip_runtime.h"
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>

// CUBLAS GEMM API
#include <hipblas.h>

// Set Cutlass debug macro to enable console printing of library errors
#define DEBUG

#if defined(WMMA)
// Conditionally include WMMA headers (CUDA 9 Preview Feature)
#include <mma.h>
#endif

// Cutlass GEMM API
#include <util/util.h>
#include <gemm/dispatch.h>
#include <gemm/epilogue_function.h>

// Test utilities
#include "util/command_line.h"
#include "util/half.h"
#include "util/matrix.h"
#include "util/timer.h"
#include "util/type_conversion.h"

// Dispatch routines to CUBLAS and CUTLASS
#include "cublas_dispatch.h"
#include "cutlass_dispatch.h"

/******************************************************************************
 * Globals, constants and typedefs
 ******************************************************************************/

using namespace cutlass;

/// CUBLAS handle
hipblasHandle_t g_cublas_handle;

/// The device-id of the current device
int g_device_id = -1;

/// The number of timing iterations to invoke
int g_timing_iterations = -1;

/// The number of randomly-sized problems to schmoo
int g_schmoo = 0;


/******************************************************************************
 * Number generation
 ******************************************************************************/

/**
 * Simple low-integer generator
 */
struct simple_gen
{
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution;

    /// Constructor
    simple_gen(int max) : distribution(max * -1, max)
    {}

    /// Functor
    int operator()()
    {
        return distribution(generator);
    }
};




/******************************************************************************
 * Test execution
 ******************************************************************************/


/**
 * Compute C = (alpha * A * B) + (beta * C)
 */
template <
    typename                    test_func_t,    ///< Test function type
    matrix_transform_t::kind_t  TransformA,     ///< Transformation op for matrix A
    matrix_transform_t::kind_t  TransformB,     ///< Transformation op for matrix B
    typename                    value_t,        ///< Multiplicand value type (matrices A and B)
    typename                    accum_t>        ///< Accumulator value type (matrix C and scalars)
bool test(
    int m,          ///< Height of C in rows
    int n,          ///< Width of C in columns
    int k,          ///< Width (height) of A (B)
    accum_t alpha,  ///< Multiplicand scalar
    accum_t beta)   ///< Addend scalar
{
    hipStream_t stream = 0;

    //
    // Initialize matrices
    //

    matrix<value_t> A(
        (TransformA == matrix_transform_t::NonTranspose) ? m : k,
        (TransformA == matrix_transform_t::NonTranspose) ? k : m);

    matrix<value_t> B(
        (TransformB == matrix_transform_t::NonTranspose) ? k : n,
        (TransformB == matrix_transform_t::NonTranspose) ? n : k);

    matrix<accum_t> C(m, n);

    // initialized matrices with small values precisely representable as integers
    simple_gen a_gen(3);
    simple_gen b_gen(5);
    A.fill_random(a_gen);
    B.fill_random(b_gen);
    C.fill_ramp(0,0);

//    // Alternatively, initialize with procedural values to simplify debugging incorrect results
//    A.fill_ramp(1,2);
//    B.fill_ramp(1,1);

    // Sync to device
    A.sync_device();
    B.sync_device();
    C.sync_device();

    CUDA_PERROR(hipPeekAtLastError());
    CUDA_PERROR(hipDeviceSynchronize());

    //
    // Run test once with debug-synchronous enabled and check result
    //

    if (!g_schmoo) printf("\n");

    test_func_t test_func;

    C.fill_ramp(0, 0);
    C.sync_device();

    hipError_t error = test_func(
        g_cublas_handle,
        m,
        n,
        k,
        A.d_data(),
        B.d_data(),
        C.d_data(),
        alpha,
        beta,
        stream,
        !g_schmoo).result;

    bool not_applicable = (error == hipErrorInvalidValue);
    bool is_failed = false;
    if (not_applicable)
    {
        printf(", NA");
    }
    else
    {
        CUDA_PERROR(error);

        // Compute reference check if wont take too long on CPU
        if ((!g_schmoo) && (m * n <= 1024 * 1024))
        {
            matrix<accum_t> ref_C(m, n);
            ref_C.fill_ramp(0, 0);
            ref_C.gemm(TransformA, TransformB, alpha, A, B, beta);
            C.sync_host();

            is_failed = (C != ref_C);

            if (!g_schmoo)
            {
                if (is_failed)
                {
                    printf("FAIL, ");
                    std::ofstream file_a("a.csv");
                    A.write_matrix(file_a);
                    std::ofstream file_b("b.csv");
                    B.write_matrix(file_b);
                    std::ofstream file_d("gemm-REF.csv");
                    ref_C.write_matrix(file_d);
                    std::ofstream file_c("gemm-GPU.csv");
                    C.write_matrix(file_c);
                }
                else
                {
                    printf("PASS, ");
                }
            }
        }
        fflush(stdout);

        //
        // Warmup and timing iterations
        //

        if (g_timing_iterations > 0)
        {
            // Warmup for 1/100 of the timing iterations (minimum of 2)
            for (int i = 0; i < __NV_STD_MAX(2, (g_timing_iterations + 99) / 100); ++i)
            {
                CUDA_PERROR(test_func(
                    g_cublas_handle,
                    m,
                    n,
                    k,
                    A.d_data(),
                    B.d_data(),
                    C.d_data(),
                    alpha,
                    beta,
                    stream,
                    false).result);
            }
        }

        // Conduct timing iterations
        double elapsed_ms = 0;
        gpu_timer timer;
        timer.start();

        for (int i = 0; i < g_timing_iterations; i++)
        {
            CUDA_PERROR(test_func(
                g_cublas_handle,
                m,
                n,
                k,
                A.d_data(),
                B.d_data(),
                C.d_data(),
                alpha,
                beta,
                stream,
                false).result);
        }

        timer.stop();
        elapsed_ms += timer.elapsed_millis();
        double avg_ms = elapsed_ms / g_timing_iterations;

        // Display performance
        if (g_timing_iterations > 0)
        {
            int64_t num_flops      = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
            double gflops_per_sec   = double(num_flops) / avg_ms / 1.0e6;

            if (g_schmoo)
            {
                if (is_failed)
                    printf("F");

                printf(", %.3f", gflops_per_sec);

                // Sleep for a few milliseconds to cool
                sleep_millis(10);
            }
            else
            {
                printf("Avg runtime: %.3f ms, total flops: %ld, GFLOP/s: %.2f\n",
                    avg_ms,
                    num_flops,
                    gflops_per_sec);
            }
            fflush(stdout);
        }
    }

    return is_failed;
}

/**
 * Compute C = (alpha * A * B) + (beta * C)
 */
template <
    math_operation_class_t     math_op,
    matrix_transform_t::kind_t TransformA,  ///< Transformation op for matrix A
    matrix_transform_t::kind_t TransformB,  ///< Transformation op for matrix B
    typename value_t,                       ///< Multiplicand value type (matrices A and B)
    typename accum_t>                       ///< Accumulator value type (matrix C and scalars)
bool test(
    int m,          ///< Height of C in rows
    int n,          ///< Width of C in columns
    int k,          ///< Width (height) of A (B)
    accum_t alpha,  ///< Multiplicand scalar
    accum_t beta)   ///< Addend scalar
{
    uint64_t flop_base = 1ull << 41;
    int max_timing_iterations = 10000;
    int min_timing_iterations = 10;

    bool test_error = false;

    // Scale the number of timing iterations with respect to problem size (if not specified on commandline)
    if ((g_timing_iterations < 0) || g_schmoo)
    {
        uint64_t num_flops = (2 * uint64_t(m) * uint64_t(n) * uint64_t(k)) + (2 * uint64_t(m) * uint64_t(n));
        g_timing_iterations = (int) ((flop_base / sizeof(value_t)) / num_flops);

        g_timing_iterations = (int) __NV_STD_MIN(max_timing_iterations, g_timing_iterations);
        g_timing_iterations = (int) __NV_STD_MAX(min_timing_iterations, g_timing_iterations);
    }

    if (g_schmoo)
    {
        printf("%d, %d, %d, %c%c, %d, %d",
            m, n, k,
            (TransformA == matrix_transform_t::NonTranspose) ? 'n' : 't',
            (TransformB == matrix_transform_t::NonTranspose) ? 'n' : 't',
            m * n,
            g_timing_iterations);
    }
    else
    {
        printf("\n------------------------------------------------------------\n");
        printf("%dx%dx%d, GEMM_%c%c, %d C elements, %d timing iterations\n",
            m, n, k,
            (TransformA == matrix_transform_t::NonTranspose) ? 'n' : 't',
            (TransformB == matrix_transform_t::NonTranspose) ? 'n' : 't',
            m * n,
            g_timing_iterations);
    }
    fflush(stdout);

    // CUBLAS
    test_error |= test<
        cublas_gemm<gemm::tiling_strategy::Unknown, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    // CUTLASS
    test_error |= test<
        cutlass_gemm_dispatch<gemm::tiling_strategy::Small, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    test_error |= test<
        cutlass_gemm_dispatch<gemm::tiling_strategy::Medium, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    test_error |= test<
        cutlass_gemm_dispatch<gemm::tiling_strategy::Large, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    test_error |= test<
        cutlass_gemm_dispatch<gemm::tiling_strategy::Tall, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    test_error |= test<
        cutlass_gemm_dispatch<gemm::tiling_strategy::Wide, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    test_error |= test<
        cutlass_gemm_dispatch<gemm::tiling_strategy::Huge, math_op, TransformA, TransformB, value_t, accum_t>,
        TransformA,
        TransformB,
        value_t,
        accum_t>(m, n, k, accum_t(alpha), accum_t(beta));

    return test_error;
}




/******************************************************************************
 * Main
 ******************************************************************************/


/**
 * Main
 */
int main(int argc, const char **argv) {
  typedef float       value_t;
  typedef float       accum_t;
  const math_operation_class_t math_op = math_operation_class_t::scalar;
  static const matrix_transform_t::kind_t TransformA = matrix_transform_t::NonTranspose;
  static const matrix_transform_t::kind_t TransformB = matrix_transform_t::NonTranspose;


    command_line args(argc, argv);
    int m           = 10240;
    int k           = 4096;
    int n           = 4096;
    float alpha     = 1.0;
    float beta      = 0.0;

    g_device_id = args.device_id;
    std::cout << g_device_id << std::endl;
    args.get_cmd_line_argument("m", m);
    args.get_cmd_line_argument("n", n);
    args.get_cmd_line_argument("k", k);
    args.get_cmd_line_argument("i", g_timing_iterations);
    args.get_cmd_line_argument("alpha", alpha);
    args.get_cmd_line_argument("beta", beta);
    args.get_cmd_line_argument("schmoo", g_schmoo);

    // Initialize cuBLAS
    if (hipblasCreate(&g_cublas_handle) != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipblasCreate() failed\n");
        exit(1);
    }

    bool test_error = false;

    if (g_schmoo)
    {
        // Run a schmoo of problem sizes
        printf("M, N, K, transpose, total_flops, timing_iterations, sol_flop/s, cublas_sol, cutlass_small_sol, cutlass_med_sol, cutlass_large_sol, cutlass_tall_sol, cutlass_wide_sol, cutlass_huge_sol\n");

        // Generate power-law distribution from [32, 16384)
        std::mt19937 gen(0);
        std::uniform_real_distribution<float> dis(5, 14);
        for (int i = 0; i < g_schmoo; ++i)
        {
        	int m = int(pow(float(2), dis(gen)));
        	int n = int(pow(float(2), dis(gen)));
        	int k = int(pow(float(2), dis(gen)));

        	// Round m and n to nearest multiple of 32 if < 128, otherwise to the nearest 128
        	m = (m < 128) ?
        			round_nearest(m, 32) :
        			round_nearest(m, 128);
        	n = (n < 128) ?
        			round_nearest(n, 32) :
        			round_nearest(n, 128);

        	// Round k to the nearest 16
            k = (sizeof(value_t) == 1) ?
                round_nearest(k, 32) :
                round_nearest(k, 16);

        	test_error |= test<math_op, TransformA, TransformB, value_t, accum_t>(
                m, n, k,
                from_float<accum_t>(alpha),
                from_float<accum_t>(beta));

        	printf("\n"); fflush(stdout);
        }
    }
    else
    {
        // Test a single GEMM problem size
        test_error |= test<math_op, TransformA, TransformB, value_t, accum_t>(
            m,
            n,
            k,
            from_float<accum_t>(alpha),
            from_float<accum_t>(beta));
    }

    // Cleanup
    hipblasDestroy(g_cublas_handle);

    return test_error;
}
